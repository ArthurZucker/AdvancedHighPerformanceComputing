#include "hip/hip_runtime.h"
/**************************************************************
This code is an implementation of the merging of two arrays
as describes in the subject
Both the sequential and parralele versions will be detailed in
order to asses the performances
***************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "merge.h"
using namespace std;
void merged_path_seq(const int *__restrict__ A,const int *__restrict__ B, int *__restrict__ M,const int a, const int b){
	int m = a+b;
	int i = 0;
	int j = 0;
	while(i+j<m){
		if(i>=a){
			M[i+j]=B[j];
			j++;}
		else if(j>=b ||A[i]<B[j]){
			M[i+j]=A[i];
			i++;}
		else{
			M[i+j]=B[j];
			j++;}
	}
}

// used shared memory
__global__ void mergeSmall_k_shared(const int *__restrict__ A,const int *__restrict__ B, int *M,const int sA, const int sB, const int sM){
    // Threads from same block share this memory 
    // In this case there is only 1 block in the call and at most 1024 threads in the block
    // Here the shared memory is dynamically allocated 
    extern __shared__ int shared[]; // dynamic. Extern allows for the host to allocate the memory 
    //__shared__ int shared[1024];  // static
    int i = threadIdx.x;            // only one thread thus Block idX not relevant 
    if(i<sM){
        // Shared: [ s1, s2 , ...., sA , sA+1, ....., sM]
        //         [   A               ,       B        ]
        if (0<=i && i<sA)shared[i] = A[i];          // threads <|A| load A[i] in shared memory 
        else if (sA<=i && i<sM)shared[i] = B[i-sA]; // threads >|A| but <|M| load B[i] in shared memory 
        // offset required, to get i == sM, i-sA = sB last index of shared and B 
        __syncthreads(); // make sure that every thread will have the data
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            // to access A[i]: shared[i]
            // to access B[i]: shared[sA+i]
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || shared[Q.y] > shared[sA+Q.x-1])){
                if(Q.x==sB || Q.y==0 || shared[Q.y-1]<=shared[sA+Q.x]){
                   if(Q.y < sA && (Q.x == sB || shared[Q.y]<=shared[sA+Q.x])){
                        M[i] = shared[Q.y];
                   }
                   else{
                        M[i] = shared[sA+Q.x];
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}
// used texture memory
__global__ void mergedSmall_k_texture(int *__restrict__ M,const int sA, const int sB, const int sM){
    int i = threadIdx.x;
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 ||
            tex1Dfetch( texture_referenceA, Q.y    ) >
            tex1Dfetch( texture_referenceB, Q.x-1  )))
            {
                if(Q.x==sB || Q.y==0 || tex1Dfetch( texture_referenceA, (Q.y-1))<=tex1Dfetch( texture_referenceB, Q.x)){
                   if(Q.y < sA && (Q.x == sB || tex1Dfetch( texture_referenceA, Q.y)<=tex1Dfetch( texture_referenceB, Q.x))){
                        M[i] = tex1Dfetch( texture_referenceA, Q.y);
                   }
                   else{
                        M[i] = tex1Dfetch( texture_referenceB, Q.x);
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}
// used ldg
__global__ void mergedSmall_k_ldg(const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ M,int sA, int sB, int sM){
    int i = threadIdx.x;
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            // __ldg intrinsic and const __restrict__ garanties the compiler that it is read only
            // thus no aliasing is done 
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || __ldg(&A[Q.y]) > __ldg(&B[Q.x-1]))){
                if(Q.x==sB || Q.y==0 || __ldg(&A[Q.y-1])<=__ldg(&B[Q.x])){
                   if(Q.y < sA && (Q.x == sB || __ldg(&A[Q.y])<=__ldg(&B[Q.x]))){
                        M[i] = __ldg(&A[Q.y]);
                   }
                   else{
                        M[i] = __ldg(&B[Q.x]);
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}
// zerocopy
__global__ void mergedSmall_k(const int *__restrict__ A,const int *__restrict__ B, int *__restrict__ M,const int sA, const int sB, const int sM){
    int i = threadIdx.x;
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || A[Q.y] > B[Q.x-1])){
                if(Q.x==sB || Q.y==0 || A[Q.y-1]<=B[Q.x]){
                   if(Q.y < sA && (Q.x == sB || A[Q.y]<=B[Q.x])){
                        M[i] = A[Q.y];
                   }
                   else{
                        M[i] = B[Q.x];
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}

__global__ void pathBig_k_naive (const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ path,const int sA,const int sB,const int sM){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || A[Q.y] > B[Q.x-1])){
                if(Q.x==sB || Q.y==0 || A[Q.y-1]<=B[Q.x]){
                   if(Q.y < sA && (Q.x == sB || A[Q.y]<=B[Q.x])){
                        path[i] = -Q.y; // 0 means I take A
                   }
                   else{
                        path[i] = Q.x; // 1 means I take B
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}

__global__ void pathBig_k_shared (const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ path,const int sA,const int sB,const int sM){
    extern __shared__ int shared[];
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (0<=i && i<sA)shared[i] = A[i];          // threads <|A| load A[i] in shared memory 
        else if (sA<=i && i<sM)shared[i] = B[i-sA]; // threads >|A| but <|M| load B[i] in shared memory 
        // offset required, to get i == sM, i-sA = sB last index of shared and B 
        __syncthreads();
    
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || A[Q.y] > B[Q.x-1])){
                if(Q.x==sB || Q.y==0 || A[Q.y-1]<=B[Q.x]){
                   if(Q.y < sA && (Q.x == sB || A[Q.y]<=B[Q.x])){
                        path[i] = -Q.y; // 0 means I take A
                   }
                   else{
                        path[i] = Q.x; // 1 means I take B
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}


__global__ void pathBig_k_naive_ldg (const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ path,const int sA,const int sB,const int sM){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || __ldg(&A[Q.y]) > __ldg(&B[Q.x-1]))){
                if(Q.x==sB || Q.y==0 || __ldg(&A[Q.y-1])<=__ldg(&B[Q.x])){
                // return Qx and Qy instantly
                   if(Q.y < sA && (Q.x == sB || __ldg(&A[Q.y])<=__ldg(&B[Q.x]))){
                        path[i] = -Q.y; // 0 means I take A
                   }
                   else{
                        path[i] = Q.x;  // 1 means I take B
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}

__global__ void    merged_Big_k_naive(const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ M, int *__restrict__ path, const int m){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i<m){
    int p = path[i];
     M[i] = p>0? B[p] : A[-p]; // if path[i] == 1 then M[i] = B[i]
    }

}

__global__ void    merged_Big_k_naive_ldg(const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ M, int *__restrict__ path, const int m){
    
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i<m){
    int p = __ldg(&path[i]);
    M[i] = p>0? __ldg(&B[p]) : __ldg(&A[-p]); // if path[i] == 1 then M[i] = B[i]
    }

}

