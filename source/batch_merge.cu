#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "batch_merge.h"

__global__ void mergeSmallBatch_k(int *__restrict__ all_M,int *M,int *all_sA, int *all_sB,int d){
    int tidx = threadIdx.x%d;
    int Qt = (threadIdx.x-tidx)/d;
    int gbx = Qt + blockIdx.x*(blockDim.x/d);
    // printf("threadIdx = %d; blockIdx = %d; tidx = %d, Qt = %d, gbx = %d, combined=%d\n",threadIdx.x,blockIdx.x,tidx,Qt,gbx,tidx+gbx*d);
    // for(int i = 0;i<6;i++){ 
    //     printf("all_size_A[%d]=%d, all_size_B[%d]=%d \n",i,sA[i],i,sB[i]);
    // }
    int blx = blockIdx.x;
    int sA = all_sA[gbx];
    int sB = all_sB[gbx];
    // printf("sB[%d]=%d\n",blx,sB[blx]);
    int *A = &all_M[gbx*d];
    int *B = &all_M[gbx*d+sA];
   
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(tidx<d){
        int2 K;
        int2 P;
        if(tidx>sA){
            K = {tidx-sA,sA};
            P = {sA,tidx-sA};
        }
        else{
            K = {0,tidx};
            P = {tidx,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            // __ldg intrinsic and const __restrict__ garanties the compiler that it is read only
            // thus no aliasing is done
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || __ldg(&A[Q.y]) > __ldg(&B[Q.x-1]))){
                if(Q.x==sB || Q.y==0 || __ldg(&A[Q.y-1])<=__ldg(&B[Q.x])){
                   if(Q.y < sA && (Q.x == sB || __ldg(&A[Q.y])<=__ldg(&B[Q.x]))){
                        M[i] = __ldg(&A[Q.y]);
                        // printf("index globale = %d, threadIdx = %d; blockIdx = %d; tidx = %d, Qt = %d, gbx = %d, __ldg(&A[%d]) = %d\n",i,threadIdx.x,blockIdx.x,tidx,Qt,gbx,Q.y,__ldg(&A[Q.y]));
                   }
                   else{
                        M[i] = __ldg(&B[Q.x]);
                        // printf("index globale = %d, threadIdx = %d; blockIdx = %d; tidx = %d, Qt = %d, gbx = %d, __ldg(&B[%d]) = %d\n",i,threadIdx.x,blockIdx.x,tidx,Qt,gbx,Q.x,__ldg(&B[Q.x]));
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}
