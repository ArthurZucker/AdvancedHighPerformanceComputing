#include "hip/hip_runtime.h"
/**************************************************************
This code is an implementation of a large number N of arrays
{Ai} and {Bi} for i = 1,...,N with |Ai|+|Bi| = d<=1024 
that merges two by two, for each i, Ai and Bi.
***************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "batch_merge.h"


__global__ void mergeSmallBatch_k(int *__restrict__ all_M,int *M,int *all_sA, int *all_sB,int d){
    
    /**
    * takes a big array @param all_M containing (Ai and Bi) suach as all_M = (A1|B1|...|AN|BN)
    * @param all_sA contains all size of different A (all_sA[0] = |A0|)
    * @param all_sB contains all size of different B (all_sB[0] = |B0|)
    * @return Nothing, each sub array is sorted
    * @note we stored sizes of Ai and Bi because |Ai|!=|Bi| and |Ai| and |Bi| not constant
    * d is the number of element that there is in the array Mi, i.e all_sA[i]+all_sB[i] = d 
    * size of M and all_M is d*N 
    */

    int tidx = threadIdx.x%d; // to know which element of the sub-array the thread treats
    int Qt = (threadIdx.x-tidx)/d;
    int gbx = Qt + blockIdx.x*(blockDim.x/d);// which array it treats

    // take the good size 
    int sA = all_sA[gbx]; 
    int sB = all_sB[gbx];
    // take the good arrays A and B
    int *A = &all_M[gbx*d];
    int *B = &all_M[gbx*d+sA];
   
    int i = blockDim.x*blockIdx.x + threadIdx.x;//global thread index  
    if(tidx<d){
        int2 K;
        int2 P;
        if(tidx>sA){
            K = {tidx-sA,sA};
            P = {sA,tidx-sA};
        }
        else{
            K = {0,tidx};
            P = {tidx,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};

            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || A[Q.y] > B[Q.x-1])){
                if(Q.x==sB || Q.y==0 || A[Q.y-1]<=B[Q.x]){
                   if(Q.y < sA && (Q.x == sB || A[Q.y]<=B[Q.x])){
                        M[i] = A[Q.y];
                   }
                   else{
                        M[i] = B[Q.x];
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}

// mergeSmallBatch using ldg 
__global__ void mergeSmallBatch_k_ldg(int *__restrict__ all_M,int *M,int *all_sA, int *all_sB,int d){
    /**
    * @see mergeSmallBatch_k(), does the same with ldg
    */

    int tidx = threadIdx.x%d;
    int Qt = (threadIdx.x-tidx)/d;
    int gbx = Qt + blockIdx.x*(blockDim.x/d);

    int sA = all_sA[gbx];
    int sB = all_sB[gbx];
    int *A = &all_M[gbx*d];
    int *B = &all_M[gbx*d+sA];
   
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(tidx<d){
        int2 K;
        int2 P;
        if(tidx>sA){
            K = {tidx-sA,sA};
            P = {sA,tidx-sA};
        }
        else{
            K = {0,tidx};
            P = {tidx,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
    
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || __ldg(&A[Q.y]) > __ldg(&B[Q.x-1]))){
                if(Q.x==sB || Q.y==0 || __ldg(&A[Q.y-1])<=__ldg(&B[Q.x])){
                   if(Q.y < sA && (Q.x == sB || __ldg(&A[Q.y])<=__ldg(&B[Q.x]))){
                        M[i] = __ldg(&A[Q.y]);
                   }
                   else{
                        M[i] = __ldg(&B[Q.x]);
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}

// mergeSmallBatch using shared memory 
__global__ void mergeSmallBatch_k_shared(int *__restrict__ all_M,int *M,int *all_sA, int *all_sB,int d){

    /**
    * @see mergeSmallBatch_k(), does the same using shared memory
    */

    int tidx = threadIdx.x%d;
    int Qt = (threadIdx.x-tidx)/d;
    int gbx = Qt + blockIdx.x*(blockDim.x/d);
    
    extern __shared__ int shared[];

    int sA = all_sA[gbx];
    int sB = all_sB[gbx];
    int *A = &all_M[gbx*d];
    int *B = &all_M[gbx*d+sA];
   
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(tidx<d){
        if (0<=tidx && tidx<sA)shared[tidx] = A[tidx];          
        else if (sA<=tidx && tidx<d)shared[tidx] = B[tidx-sA];
        __syncthreads();
        int2 K;
        int2 P;
        if(tidx>sA){
            K = {tidx-sA,sA};
            P = {sA,tidx-sA};
        }
        else{
            K = {0,tidx};
            P = {tidx,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            // to access A[i]: shared[i]
            // to access B[i]: shared[sA+i]
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || shared[Q.y] > shared[sA+Q.x-1])){
                if(Q.x==sB || Q.y==0 || shared[Q.y-1]<=shared[sA+Q.x]){
                   if(Q.y < sA && (Q.x == sB || shared[Q.y]<=shared[sA+Q.x])){
                        M[i] = shared[Q.y];
                   }
                   else{
                        M[i] = shared[sA+Q.x];
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}
