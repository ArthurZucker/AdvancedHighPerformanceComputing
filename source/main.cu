#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "merge.h"
#include "batch_merge.h"
#include "utils.h"
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))
using namespace std;
#define TEXTURE 0 //set to 0 to use normal memory, else it will use texture memory for A and B
texture <int> texture_referenceA ;
texture <int> texture_referenceB ;
#define QUESTION 4
#define INFO 0
/*
TO DO :
 - implement using ldg  avec restricted__  et int4 qui contient 4 int, read only memory
     const int* __restrict__  A
 - mergeBig_k
 - pathBig_k

*/
int main(int argc, char* argv[]) {
    hipDeviceReset();
    //___________ Basic initialisation ___________
	srand((unsigned int)time(NULL));
	int nDevices;
	hipGetDeviceCount(&nDevices);
    int Tmax;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        #if INFO == 1
		printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
        printf("Max number of threads per multiprocessor : %d\n",prop.maxThreadsPerMultiProcessor);
        printf("Number of multiprocessors on device : %d\n",prop.multiProcessorCount);
        printf("Amount of Shared mem available for int : %d\n",prop.sharedMemPerMultiprocessor/sizeof(int));
        printf("Max running threads : %d\n",prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount);
        #endif
        Tmax = prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount;
	}
    //Tmax =1024;
	hipSetDevice(0);
    testCUDA(hipSetDeviceFlags(hipDeviceMapHost));
    //____________________________________________

    //___________ Variable declaration ___________
    int sizeA,sizeB;
    if (argc < 2) {sizeA = rand()%1024;sizeB = rand()%(1024-sizeA);} // If no arguments are provided, set random sizes
    else if(argc == 2){sizeA=atoi(argv[1]);sizeB=atoi(argv[1]);}
    else{sizeA=atoi(argv[1]);sizeB=atoi(argv[2]);}
    int sizeM = sizeA+sizeB;
    printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
    int *hostA,*thostA,*hostB,*thostB,*hostM,*hA,*hB,*hM;
    int *seqM = (int *) malloc(sizeM*sizeof(int));
    int *A = (int *) malloc(sizeA*sizeof(int));
    int *B = (int *) malloc(sizeB*sizeof(int));
    int *M = (int *) malloc(sizeM*sizeof(int));
    A[0]=rand()%20;
    B[0]=rand()%20;
    for(int i=1;i<sizeA;i++){A[i]=A[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){B[i]=B[i-1]+rand()%20+1;}


    //___________ call kernels ___________________
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
    float TimeVar=0;



    #if QUESTION == 1
    //___________ TO DO: explain texture memory ___________
    testCUDA(hipMalloc((void **)&thostA,sizeA*sizeof(int)));
    testCUDA(hipMalloc((void **)&thostB,sizeB*sizeof(int)));

    testCUDA(hipMemcpy(thostA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(thostB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));

    testCUDA (hipBindTexture(0,texture_referenceA, thostA,sizeA*sizeof(int)));
    testCUDA (hipBindTexture(0,texture_referenceB, thostB,sizeB*sizeof(int)));
    //____________________________________________
    // zero copy
    testCUDA(hipHostAlloc(&hostA,sizeA*sizeof(int),hipHostMallocMapped)); //hipHostMallocWriteCombined
    testCUDA(hipHostAlloc(&hostB,sizeB*sizeof(int),hipHostMallocMapped));
    hostA[0]=rand()%20;
    hostB[0]=rand()%20;
    for(int i=1;i<sizeA;i++){hostA[i]=hostA[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){hostB[i]=hostB[i-1]+rand()%20+1;}

    // WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read
    // efficiently by most CPUs. WC memory is a good option for buffers that will be written by the CPU and read by the device
    // via mapped pinned memory or host->device transfers.

    testCUDA(hipHostAlloc(&hostM,sizeM*sizeof(int),hipHostMallocMapped)); // in order to do zero copy
    /*testCUDA(hipHostGetDevicePointer((void **)&pM, (void *) hostM,0));
    testCUDA(hipHostGetDevicePointer((void **)&pA, (void *) hostA,0));
    testCUDA(hipHostGetDevicePointer((void **)&pB, (void *) hostB,0));
    */
    //_______________ Sequential _________________
    printf("_______________ Sequential _________________\n");
    clock_t begin = clock();
    merged_path_seq(A,B,seqM,sizeA,sizeB);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("elapsed time : %f ms\n",time_spent*1000);
    cout<<"Check sorted : "<<is_sorted(seqM,sizeM)<<endl;
    //____________________________________________


    
    //____________________________________________

    //___________ Shared _________________________
    printf("________________ Shared ___________________\n");
    testCUDA(hipEventRecord(start));
    mergeSmall_k_shared<<<1,sizeM,sizeM*sizeof(int)>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    //mergeSmall_k_shared<<<1,sizeM>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ texture ________________________
    printf("________________ Texture ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k_texture<<<1,1024>>>(hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ zerocopy _______________________
    printf("_______________ zero copy ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ LDG ____________________________
    printf("_____________________ LDG ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k_ldg<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________
    for(int i=1;i<sizeA;i++){hostM[i]=0;}
    
    #endif
    #if QUESTION==2
    //___________ MergeBig _______________________
    printf("__________________ Path big normal __________________\n");
    int *__restrict__ path;
    int nb_threads = 5;
    int nb_blocks = (sizeM+nb_threads-1)/nb_threads;
    if(sizeM<1024) nb_blocks=1024;
    nb_blocks = 2;
    testCUDA(hipMalloc((void **)&hA,sizeA*sizeof(int)));
    testCUDA(hipMalloc((void **)&hB,sizeB*sizeof(int)));
    testCUDA(hipMalloc((void **)&hM,sizeM*sizeof(int)));

    testCUDA(hipMemcpy(hA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(hB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));

    testCUDA(hipMalloc((void **)&path,2*(nb_blocks+1)*sizeof(int)));
    testCUDA(hipEventRecord(start,0));
    pathBig_k<<<nb_blocks,nb_threads>>>(hA,hB,path,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
  
    //___________ Path Big _______________________
    printf("__________________ Merg big normal _________________\n");
    testCUDA(hipEventRecord(start,0));
    merged_Big_k<<<nb_blocks,nb_threads>>>(hA,hB,hM,path,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    testCUDA(hipMemcpy(M, hM, sizeB*sizeof(int), hipMemcpyDeviceToHost));
    cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;
    //print_t(hostM,sizeM);
    //____________________________________________
    #endif
  
    //___________ MergeBig _______________________
    // printf("__________________ Path big sans shared + ldg __________________\n");
    // testCUDA(hipEventRecord(start,0));
    // pathBig_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,path,sizeA,sizeB,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
  
    //___________ Path Big _______________________
    // printf("__________________ Merg big sans shared + ldg _________________\n");
    // testCUDA(hipEventRecord(start,0));
    // merged_Big_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,hostM,path,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    // cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________
    //___________ Cleaning up ____________________
    #if QUESTION == 1
    testCUDA(hipUnbindTexture ( texture_referenceA ));
    testCUDA(hipUnbindTexture ( texture_referenceB ));
    hipFree(thostA);
    hipFree(thostB);
    testCUDA(hipHostFree(hostA));
    testCUDA(hipHostFree(hostB));
    testCUDA(hipHostFree(hostM));
    #endif
    free(A);
    free(B);
    free(M);
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
    //____________________________________________
    #if QUESTION==4
    //__________________________ Batch merge part __________________________
    // L’objectif est simplement de répartir les block de manière intelligente 
    // sur l’ensemble des calculs Ai + Bi = Mi .
    int N = 100; //si trop gros on pet pas allouer sur le gpu (je crois)
    int d = 306;
    // int sizeA,sizeB,sizeM;

    // int** all_A = (int**)malloc(N*sizeof(int*));
    // int** all_B = (int**)malloc(N*sizeof(int*));
    // int** all_M = (int**)malloc(N*sizeof(int*));
    // int* all_size_A = (int*)malloc(N*sizeof(int));
    // int* all_size_B = (int*)malloc(N*sizeof(int));
    // int* all_size_M = (int*)malloc(N*sizeof(int));

    int** all_A;
    int** all_B; 
    int** all_M;
    int* all_size_A;
    int* all_size_B;
    // int* all_size_M;
    testCUDA(hipHostAlloc(&all_A,N*sizeof(int*),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_B,N*sizeof(int*),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_M,N*sizeof(int*),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_size_A,N*sizeof(int),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_size_B,N*sizeof(int),hipHostMallocMapped));
    // testCUDA(hipHostAlloc(&all_size_M,N*sizeof(int),hipHostMallocMapped));

    printf("_______ Initialisation___________\n");
    for(int i = 0;i<N;i++){
        // printf("i = %d\n",i);
        sizeA = rand()%d;
        sizeB = (d-sizeA);
        sizeM = sizeA+sizeB;
        // printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
        all_size_A[i] = sizeA;
        all_size_B[i] = sizeB;
        // all_size_M[i] = sizeM;

        // all_A[i] = (int *) malloc(sizeA*sizeof(int));
        // all_B[i] = (int *) malloc(sizeB*sizeof(int));
        // all_M[i] = (int *) malloc(sizeM*sizeof(int));
        testCUDA(hipHostAlloc(&all_A[i],sizeA*sizeof(int),hipHostMallocMapped));
        testCUDA(hipHostAlloc(&all_B[i],sizeB*sizeof(int),hipHostMallocMapped));
        testCUDA(hipHostAlloc(&all_M[i],sizeM*sizeof(int),hipHostMallocMapped));

        all_A[i][0]=rand()%20;
        all_B[i][0]=rand()%20;
        for(int j=1;j<sizeA;j++){all_A[i][j]=all_A[i][j-1]+rand()%20+1;}
        for(int j=1;j<sizeB;j++){all_B[i][j]=all_B[i][j-1]+rand()%20+1;}
    
    }
    // for(int i = 0;i<N;i++){
    //     printf("size A[%d] = %d\n",i,all_size_A[i]);
    //     printf("size B[%d] = %d\n",i,all_size_B[i]);
    //     //printf("size M[%d] = %d\n",i,all_size_M[i]);
    //     for(int j = 0; j< all_size_A[i];j++){
    //         printf("all_A[%d] = %d\n",j,all_A[i][j]);
    //     }
    // }

    printf("_______ Début de la fonction___________\n");
    int numBlocks = 2; 
    int threadsPerBlock = 512;
    testCUDA(hipEventRecord(start));
    mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(all_A,all_B,all_M,all_size_A,all_size_B,d);
    testCUDA(hipEventRecord(stop));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    #endif
    // printf("_______ Check résultats___________\n");
    // for(int i = 0;i<1;i++){
    //     for(int j = 0;j<d;j++){
    //         printf("M[%d][%d]=%d\n",i,j,all_M[i][j]);
    //     }
    // }
    // for(int i = 0;i<N;i++){
    //     //printf("%d\n",i);
    //     cout<<"Check sorted : "<<is_sorted(all_M[i],d)<<endl;
    // }
    
    
    //for(int i = 0;i<N;i++){free(all_A[i]);free(all_B[i]);free(all_M[i]);}
    // free(all_A);
    // free(all_B);
    // free(all_M);
    // free(all_size_A);
    // free(all_size_B);
    // free(all_size_M);
    // for(int i = 0;i<N;i++){testCUDA(hipHostFree(all_A[i]));testCUDA(hipHostFree(all_B[i]));testCUDA(hipHostFree(all_M[i]));}
    // testCUDA(hipHostFree(all_A));
    // testCUDA(hipHostFree(all_B));
    // testCUDA(hipHostFree(all_M));
    // testCUDA(hipHostFree(all_size_A));
    // testCUDA(hipHostFree(all_size_B));
    
    // testCUDA(hipEventDestroy(start));
	// testCUDA(hipEventDestroy(stop));
	return 0;
}
