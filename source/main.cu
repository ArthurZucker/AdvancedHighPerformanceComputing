#include "hip/hip_runtime.h"
/****************************************************************************
 * Copyright (C) 2020 by Arthur Zucker @ Apavou Clément                     *
 ****************************************************************************/

/**
 * @file main.cu
 * @author Arthur Zucker & Clément Apavou  
 * @date 12 Dec 2020
 * @brief Main file used to produce results for each questions
 *
 * In this porject, we tackled the MERGE SORT problem on GPU
 * using CUDA. We answered questions from a subject. If you want to 
 * see the original Merge sort articles, 
 * @see https://www.researchgate.net/profile/Oded-Green/publication/254462662_GPU_merge_path_a_GPU_merging_algorithm/links/543eeaa00cf2e76f02244884/GPU-merge-path-a-GPU-merging-algorithm.pdf
 * @see https://arxiv.org/pdf/1406.2628.pdf 
 */


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "merge.h"
#include "batch_merge.h"
#include "utils.h"
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))
using namespace std;
texture <int> texture_referenceA ;
texture <int> texture_referenceB ;
#define QUESTION 1  /**< Choose from {1,2,3,4,5} depending on the question */
#define INFO 0      /**< Set to 1 if you need to see GPU infromations. */


int main(int argc, char* argv[]) {
    //___________ Basic initialisation ___________
	srand((unsigned int)time(NULL));
	int nDevices;
	hipGetDeviceCount(&nDevices);
    // int Tmax;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        #if INFO == 1
            printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
            printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
            printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
            printf("Max number of threads per multiprocessor : %d\n",prop.maxThreadsPerMultiProcessor);
            printf("Number of multiprocessors on device : %d\n",prop.multiProcessorCount);
            printf("Amount of Shared mem available for int : %d\n",prop.sharedMemPerMultiprocessor/sizeof(int));
            printf("Max running threads : %d\n",prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount);
        #endif
        // Tmax = prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount;
	}
    //Tmax =1024;
	hipSetDevice(0);
    testCUDA(hipSetDeviceFlags(hipDeviceMapHost));
    //____________________________________________

    //___________ Initialising size of arrays  ___________
    #if QUESTION==2 || QUESTION ==1
        int sizeA,sizeB;
        if (argc < 2) {sizeA = rand()%1024;sizeB = rand()%(1024-sizeA);} // If no arguments are provided, set random sizes
        else if(argc == 2){sizeA=atoi(argv[1]);sizeB=atoi(argv[1]);}
        else{sizeA=atoi(argv[1]);sizeB=atoi(argv[2]);}
        int sizeM = sizeA+sizeB;
        printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
        #if QUESTION ==1
            int *hostA,*thostA,*hostB,*thostB,*hostM,*thostM;
        #endif
        int *seqM = (int *) malloc(sizeM*sizeof(int));
        int *A = (int *) malloc(sizeA*sizeof(int));
        int *B = (int *) malloc(sizeB*sizeof(int));
        int *M = (int *) malloc(sizeM*sizeof(int));
        A[0]=rand()%20;
        B[0]=rand()%20;
        for(int i=1;i<sizeA;i++){A[i]=A[i-1]+rand()%20+1;}
        for(int i=1;i<sizeB;i++){B[i]=B[i-1]+rand()%20+1;}
    #endif
    #if QUESTION == 5
        int sizeM;
        if (argc < 2) {sizeM = rand()%1024;} 
        if (argc == 2) {sizeM=atoi(argv[1]);} // If no arguments are provided, set random sizes
        printf("|M| = %d\n",sizeM);
    #endif
    //___________________________ Useful time stamps _________________________________
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
    float TimeVar=0;


    //___________________________ Question 1 _________________________________
    #if QUESTION == 1
        FILE *f = fopen("../results/results1.csv", "w"); 
        fprintf(f, "type,memory,time\n");
        // Copy 
        testCUDA(hipMalloc((void **)&thostA,sizeA*sizeof(int)));
        testCUDA(hipMalloc((void **)&thostB,sizeB*sizeof(int)));
        testCUDA(hipMalloc((void **)&thostM,sizeM*sizeof(int)));

        testCUDA(hipMemcpy(thostA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
        testCUDA(hipMemcpy(thostB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));
        // texture memory
        testCUDA (hipBindTexture(0,texture_referenceA, thostA,sizeA*sizeof(int)));
        testCUDA (hipBindTexture(0,texture_referenceB, thostB,sizeB*sizeof(int)));
        //____________________________________________
        // Zero copy
        testCUDA(hipHostAlloc(&hostA,sizeA*sizeof(int),hipHostMallocMapped)); //hipHostMallocWriteCombined
        testCUDA(hipHostAlloc(&hostB,sizeB*sizeof(int),hipHostMallocMapped));
        hostA[0]=rand()%20;
        hostB[0]=rand()%20;
        for(int i=1;i<sizeA;i++){hostA[i]=hostA[i-1]+rand()%20+1;}
        for(int i=1;i<sizeB;i++){hostB[i]=hostB[i-1]+rand()%20+1;}

        // WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read
        // efficiently by most CPUs. WC memory is a good option for buffers that will be written by the CPU and read by the device
        // via mapped pinned memory or host->device transfers.

        testCUDA(hipHostAlloc(&hostM,sizeM*sizeof(int),hipHostMallocMapped)); // in order to do zero copy
 
        //_______________ Sequential _________________
        printf("_______________ Sequential _________________\n");
        clock_t begin = clock();
        merged_path_seq(A,B,seqM,sizeA,sizeB);
        clock_t end = clock();
        double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
        printf("elapsed time : %f ms\n",time_spent*1000);
        fprintf(f, "Sequential,CPU,%f\n",time_spent*1000);
        cout<<"Check sorted : "<<is_sorted(seqM,sizeM)<<endl;
        //____________________________________________


        //_____________________________ Zero copy ______________________________________________________________
        printf("__________________________Zero copy________________________________\n");
        printf("_______________ Zero copy Normal ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Normal,ZeroCpy,%f\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
        //____________________________________________

        for(int i=0;i<sizeM;i++){hostM[i]=0;}

        //___________ Zero copy Shared _________________________
        printf("________________ Zero copy Shared ___________________\n");
        testCUDA(hipEventRecord(start));
        mergeSmall_k_shared<<<1,sizeM,sizeM*sizeof(int)>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Shared,ZeroCpy,%f\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
        //____________________________________________

        for(int i=0;i<sizeM;i++){hostM[i]=0;}

        //___________ Zero copy LDG ____________________________
        printf("_____________________ Zero copy LDG ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k_ldg<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Ldg,ZeroCpy,%f\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
        //____________________________________________

        for(int i=0;i<sizeM;i++){hostM[i]=0;}

        //___________ Texture ________________________
        printf("________________ Texture ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Texture,ZeroCpy,%f\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
        //____________________________________________
        
        //_____________________________ Copy ______________________________________________________________
        printf("__________________________Copy________________________________\n");
        printf("_______________copy Normal ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k<<<1,1024>>>(thostA,thostB,thostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Normal,MemCpy,%f\n",TimeVar);
        testCUDA(hipMemcpy(M, thostM, sizeM*sizeof(int), hipMemcpyDeviceToHost)); // retrieve M on the device
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;
        //____________________________________________

        for(int i=0;i<sizeM;i++){M[i]=0;}

        //___________ copy Shared _________________________
        printf("________________copy Shared ___________________\n");
        testCUDA(hipEventRecord(start));
        mergeSmall_k_shared<<<1,sizeM,sizeM*sizeof(int)>>>(thostA,thostB,thostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Shared,MemCpy,%f\n",TimeVar);
        testCUDA(hipMemcpy(M, thostM, sizeM*sizeof(int), hipMemcpyDeviceToHost));
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;
        //____________________________________________

        for(int i=0;i<sizeM;i++){M[i]=0;}

        //___________ copy LDG ____________________________
        printf("_____________________copy LDG ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k_ldg<<<1,1024>>>(thostA,thostB,thostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Ldg,MemCpy,%f\n",TimeVar);
        testCUDA(hipMemcpy(M, thostM, sizeM*sizeof(int), hipMemcpyDeviceToHost));
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;
        //____________________________________________

        testCUDA(hipUnbindTexture ( texture_referenceA ));
        testCUDA(hipUnbindTexture ( texture_referenceB ));
        hipFree(thostA);
        hipFree(thostB);
        hipFree(thostM);
        testCUDA(hipHostFree(hostA));
        testCUDA(hipHostFree(hostB));
        testCUDA(hipHostFree(hostM));
        fclose(f); 
    #endif

    //___________________________ Question 2_________________________________
    #if QUESTION==2
        FILE *f = fopen("../results/results2.csv", "w"); 
        fprintf(f, "Kernel,type,memory,time\n");
        int *__restrict__ path;
        int nb_threads = 128;
        int nb_blocks = (sizeM+nb_threads-1)/nb_threads;
        //if(sizeM<1024) nb_blocks=1024;
        int *hA,*hB,*hM;
        testCUDA(hipMalloc((void **)&hA,sizeA*sizeof(int)));
        testCUDA(hipMalloc((void **)&hB,sizeB*sizeof(int)));
        testCUDA(hipMalloc((void **)&hM,sizeM*sizeof(int)));
        testCUDA(hipMemcpy(hA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
        testCUDA(hipMemcpy(hB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));
        testCUDA(hipMalloc((void **)&path,2*(nb_blocks+1)*sizeof(int)));

        //_________________________ Path and Merge global copy ___________________
        printf("__________________ Path big normal __________________\n");
        testCUDA(hipEventRecord(start,0));
        pathBig_k<<<nb_blocks,nb_threads>>>(hA,hB,path,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Path,Normal,MemCopy,%f\n",TimeVar);
        //____________________________________________
        printf("__________________ Merg big normal _________________\n");
        testCUDA(hipEventRecord(start,0));
        merged_Big_k<<<nb_blocks,nb_threads>>>(hA,hB,hM,path,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Merge,Normal,MemCopy,%f\n",TimeVar);
        testCUDA(hipMemcpy(M, hM, sizeM*sizeof(int), hipMemcpyDeviceToHost));
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;

        //_________________________ Path and Merge zero copy ___________________
        int *hzA,*hzB,*hzM;
        testCUDA(hipHostAlloc(&hzA,sizeA*sizeof(int),hipHostMallocMapped)); //hipHostMallocWriteCombined
        testCUDA(hipHostAlloc(&hzB,sizeB*sizeof(int),hipHostMallocMapped));
        hzA[0]=rand()%20;
        hzB[0]=rand()%20;
        for(int i=1;i<sizeA;i++){hzA[i]=hzA[i-1]+rand()%20+1;}
        for(int i=1;i<sizeB;i++){hzB[i]=hzB[i-1]+rand()%20+1;}
        testCUDA(hipHostAlloc(&hzM,sizeM*sizeof(int),hipHostMallocMapped));

        printf("__________________ Path big zero copy __________________\n");
        testCUDA(hipEventRecord(start,0));
        pathBig_k<<<nb_blocks,nb_threads>>>(hzA,hzB,path,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Path,Normal,ZeroCpy,%f\n",TimeVar);
        //____________________________________________
        printf("__________________ Merg big zero copy _________________\n");
        testCUDA(hipEventRecord(start,0));
        merged_Big_k<<<nb_blocks,nb_threads>>>(hzA,hzB,hzM,path,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Merge,Normal,ZeroCpy,%f\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hzM,sizeM)<<endl;

        //_________________________ Path and Merge naive ___________________
        printf("__________________ Path big for naive merge __________________\n");
        testCUDA(hipEventRecord(start,0));
        pathBig_k<<<nb_blocks,1>>>(hA,hB,path,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Path,Naive,MemCpy,%f\n",TimeVar);
        //____________________________________________
        printf("__________________ Merg big NAIVE_________________\n");
        testCUDA(hipEventRecord(start,0));
        merged_Big_k_naive<<<nb_blocks,1>>>(hA,hB,hM,path,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Path,Naive,MemCpy,%f\n",TimeVar);
        testCUDA(hipMemcpy(M, hM, sizeM*sizeof(int), hipMemcpyDeviceToHost));
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;

        //_________________________ Path and Merge naive zero copy___________________
        printf("__________________ Path big for naive merge zero copy___________________\n");
        testCUDA(hipEventRecord(start,0));
        pathBig_k<<<nb_blocks,1>>>(hzA,hzB,path,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Path,Naive,ZeroCpy,%f\n",TimeVar);
        //____________________________________________
        printf("__________________ Merg big NAIVE zero copy__________________\n");
        testCUDA(hipEventRecord(start,0));
        merged_Big_k_naive<<<nb_blocks,1>>>(hzA,hzB,hzM,path,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Merge,Naive,ZeroCpy,%f\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hzM,sizeM)<<endl;
        fclose(f); 
    #endif

    //___________________________ Question 3_________________________________
    #if QUESTION==3
        int *__restrict__ hD;
        int *__restrict__ hsD;
        int *D  ;
        int *sD ;
        int padding = 0;
        //int nb_threads = 128; // changing it might be smart
        //int nb_blocks = (sizeM+nb_threads-1)/nb_threads;
        printf("__________________ sort M __________________\n");
        int threads_per_blocks = 128;
        FILE *f = fopen("../results/results3.csv", "w"); 
        fprintf(f, "d,time\n");
        for(int d=2;d<262144*2*2;d*=4){
            testCUDA(hipMalloc((void **)&hsD,d*sizeof(int)));
            testCUDA(hipMalloc((void **)&hD ,d*sizeof(int)));
            
            //code to launch on a size != than a power of 2
            if(d != 0 && (d & (d-1)) == 0){
                //printf("|M| is a power of 2\n");
                D  = (int *) malloc(d*sizeof(int));
                sD = (int *) malloc(d*sizeof(int));
                for(int i=0;i<d;i++){D[i]=rand()%d*50+1;}
            }
            else{
                //printf("|M| was not a power of 2, it will be changed\n");
                int power = 1;
                while(power < d) power*=2;
                //printf("new |M| with padding : %d\n",power);
                D  = (int *) malloc(power*sizeof(int));
                sD = (int *) malloc(power*sizeof(int));
                for(int i=0;i<d;i++){D[i]=rand()%d*5+1;}
                for(int i = d;i<power;i++){D[i] = ( int) -1 >> 1;}
                padding = power-d;
                d = power;
            }
            
            // printf("Assigning M\n");  
            testCUDA(hipMemcpy(hD, D, d*sizeof(int), hipMemcpyHostToDevice));
            testCUDA(hipEventRecord(start,0));
            sort_array(hD,hsD,d,threads_per_blocks);
            testCUDA(hipEventRecord(stop,0));
            testCUDA(hipEventSynchronize(stop));
            testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
            printf("d = %10d | t =  %4.10f ms | ",d,TimeVar);
            fprintf(f, "%d,%f\n",d,TimeVar);
            testCUDA(hipMemcpy(sD, hsD, d*sizeof(int), hipMemcpyDeviceToHost));
            cout<<" Sorted : "<<is_sorted(sD,d);
            //____________________Compare with qsort ________________________
            clock_t begin = clock();
            qsort(D, d, sizeof(int), cmpfunc);
            clock_t end = clock();
            double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
            printf("\tquicksort t = %f ms | ",time_spent*1000);
            int sorted = 1;
            for(int i=0;i<d;i++) {
                if(D[i]!=sD[i]){
                    printf("ERROR    i=%d : %d != %d\n",i,D[i],sD[i]);
                    sorted = 0;
                    break;
                }
            }
            if(sorted) printf("arrays are equal\n");
            hipFree(hD);
            hipFree(hsD);
            free(D);
            free(sD);
        }
        fclose(f); 
    #endif
        
    //___________________________ Question 4_________________________________
    #if QUESTION==4
        FILE *f = fopen("../results/results4.csv", "w"); 
        fprintf(f, "type,memory,time\n");
        // N arrays containing Ai and Bi such as |Ai| + |Bi| = d
        // N arrays of size d
        int N = 10000; // max 1000000
        int d = 500; 
        if (argc > 2) {
            N = atoi(argv[1]);
            d = atoi(argv[2]);
            if(d>1024) d=1024; 
        } // If no arguments are provided, set random sizes
        printf("N = %d | d = %d\n",N,d);
        
        
        // ________________________________________Zero Copy______________________________________________ 

        printf("_______________________________Zero copy____________________________________\n");
        int* host_all_M;
        int* host_all_STM;
        int* host_all_size_A;
        int* host_all_size_B;

        // allocation on the device for save all size of Ai and Bi 
        // we choose a 1D representation,  we stocked Ai and Bi in one table M : M = (A1|B1|...|AN|BN) 
        testCUDA(hipHostAlloc(&host_all_size_A,N*sizeof(int),hipHostMallocMapped));
        testCUDA(hipHostAlloc(&host_all_size_B,N*sizeof(int),hipHostMallocMapped));

        // Initialisation of size Ai and Bi such as |Ai| + |Bi| = d 
        int size_all_A=0;
        int size_all_B=0;
        int sizeA;
        int sizeB;
        for(int i = 0;i<N;i++){ 
            sizeA = rand()%d+1;
            sizeB = (d-sizeA);
            host_all_size_A[i] = sizeA;
            host_all_size_B[i] = sizeB;
            size_all_A += sizeA;
            size_all_B +=sizeB;
        }

        // we stocked Ai and Bi in one table M : M = (A1|B1|...|AN|BN) 
        // allocation on device for M and STM of size N*d (N arrays of size d)
        // M will contains N arrays of Ai and Bi not sorted  
        // STM (Sorted M) will contains Mi sorted i.e Ai and Bi merge and sort 
        testCUDA(hipHostAlloc(&host_all_STM,N*d*sizeof(int),hipHostMallocMapped));    
        testCUDA(hipHostAlloc(&host_all_M,N*d*sizeof(int),hipHostMallocMapped));  

        // Start initialisation of the first arrays A0 and B0
        if(host_all_size_A[0]!=0){
            host_all_M[0]=rand()%20+1;
            for(int j = 1;j<host_all_size_A[0];j++){
                host_all_M[j]=host_all_M[j-1]+rand()%20+1;
            }
        }
        if(host_all_size_B[0]!=0){
            host_all_M[host_all_size_A[0]]=rand()%20+1;
            for(int j = host_all_size_A[0]+1;j<host_all_size_B[0]+host_all_size_A[0];j++){
                host_all_M[j]=host_all_M[j-1]+rand()%20+1;
            }
        }
            
        // Initialisation of all arrays 
        int tmp_A=host_all_size_A[0];
        int tmp_B=host_all_size_B[0];
        for(int i = 1;i<N;i++){ 
            if(host_all_size_A[i]!=0){
                host_all_M[tmp_A+tmp_B]=rand()%20+1;
                for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+host_all_size_A[i];j++){
                    host_all_M[j]=host_all_M[j-1]+rand()%20+1;
                }
                    tmp_A+= host_all_size_A[i];
            
            }
            if(host_all_size_B[i]!=0){
                host_all_M[tmp_A+tmp_B]=rand()%20+1;
                for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+host_all_size_B[i];j++){
                    host_all_M[j]=host_all_M[j-1]+rand()%20+1;
                }
                tmp_B+= host_all_size_B[i];
            }
        }
        
        printf("_________________ LDG_____________________\n");

        int numBlocks = N; //big number
        int threadsPerBlock = d; // multiple of d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k_ldg<<<numBlocks,threadsPerBlock>>>(host_all_M,host_all_STM,host_all_size_A,host_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Ldg,ZeroCpy,%f\n",TimeVar);
        // _______________Check results_______________
        int all_sorted=1;
        int sorted;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&host_all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        for(int i=0;i<d;i++){host_all_STM[i]=0;}

        printf("_________________ Shared_____________________\n");

        numBlocks = N; //big number
        threadsPerBlock = d; // multiple de d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k_shared<<<numBlocks,threadsPerBlock,d*sizeof(int)>>>(host_all_M,host_all_STM,host_all_size_A,host_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Shared,ZeroCpy,%f\n",TimeVar);
        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&host_all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        for(int i=0;i<d;i++){host_all_STM[i]=0;}

        printf("_________________ Normal_____________________\n");

        numBlocks = N; //big number
        threadsPerBlock = d; // multiple de d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(host_all_M,host_all_STM,host_all_size_A,host_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Normal,ZeroCpy,%f\n",TimeVar);
        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&host_all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        // ________________________________________Copy______________________________________________ 

        printf("__________________________________Copy_______________________________________\n");

        int* all_M = (int *) malloc(N*d*sizeof(int));
        int* all_STM = (int *) malloc(N*d*sizeof(int));
        int* all_size_A = (int *) malloc(N*sizeof(int));
        int* all_size_B = (int *) malloc(N*sizeof(int));
        int* h_all_M;
        int* h_all_STM;
        int* h_all_size_A;
        int* h_all_size_B;

        // allocation on device for save size
        testCUDA(hipMalloc((void **)&h_all_size_A,N*sizeof(int)));
        testCUDA(hipMalloc((void **)&h_all_size_B,N*sizeof(int)));

        // Initialisation size
        size_all_A=0;
        size_all_B=0;
        for(int i = 0;i<N;i++){ 
            sizeA = rand()%d+1;
            sizeB = (d-sizeA);
            all_size_A[i] = sizeA;
            all_size_B[i] = sizeB;
            size_all_A += sizeA;
            size_all_B +=sizeB;
        }

        // copy of all size on device
        testCUDA(hipMemcpy(h_all_size_A, all_size_A, N*sizeof(int), hipMemcpyHostToDevice));
        testCUDA(hipMemcpy(h_all_size_B, all_size_B, N*sizeof(int), hipMemcpyHostToDevice));
        
        // allocation on device of M and STM
        testCUDA(hipMalloc((void **)&h_all_M,N*d*sizeof(int)));
        testCUDA(hipMalloc((void **)&h_all_STM,N*d*sizeof(int)));

        // Start initialisation of the first arrays A0 and B0
        if(all_size_A[0]!=0){
            all_M[0]=rand()%20+1;
            for(int j = 1;j<all_size_A[0];j++){
                all_M[j]=all_M[j-1]+rand()%20+1;
            }
        }
        if(all_size_B[0]!=0){
            all_M[all_size_A[0]]=rand()%20+1;
            for(int j = all_size_A[0]+1;j<all_size_B[0]+all_size_A[0];j++){
                all_M[j]=all_M[j-1]+rand()%20+1;
            }
        }
        tmp_A=all_size_A[0];
        tmp_B=all_size_B[0];

        // Initialisation of all arrays 
        for(int i = 1;i<N;i++){ 
            if(all_size_A[i]!=0){
                all_M[tmp_A+tmp_B]=rand()%20+1;
                for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_A[i];j++){
                    all_M[j]=all_M[j-1]+rand()%20+1;
                }
                tmp_A+= all_size_A[i];
        
            }
            if(all_size_B[i]!=0){
                all_M[tmp_A+tmp_B]=rand()%20+1;
                for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_B[i];j++){
                    all_M[j]=all_M[j-1]+rand()%20+1;
                }
                tmp_B+= all_size_B[i];
            }
        }
        // copy all_M on h_all_M on device
        testCUDA(hipMemcpy(h_all_M, all_M, N*d*sizeof(int), hipMemcpyHostToDevice));

        printf("_________________ LDG_____________________\n");

        numBlocks = N; //big number
        threadsPerBlock = d; // multiple de d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k_ldg<<<numBlocks,threadsPerBlock>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Ldg,Memcpy,%f\n",TimeVar);
        // retrieve STM on device
        testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }

        for(int i=0;i<d;i++){all_STM[i]=0;}

        printf("_________________ Shared_____________________\n");

        numBlocks = N; //big number
        threadsPerBlock = d; // multiple de d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k_shared<<<numBlocks,threadsPerBlock,d*sizeof(int)>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Shared,Memcpy,%f\n",TimeVar);
        testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        for(int i=0;i<d;i++){all_STM[i]=0;}

        printf("_________________ Normal_____________________\n");

        numBlocks = N; //big number
        threadsPerBlock = d; // multiple of d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        fprintf(f, "Normal,Memcpy,%f\n",TimeVar);
        testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        // test on quicksort sequential to compare 
        printf("______________________________Quicksort sequential___________________________\n");

        clock_t begin = clock();
        for(int i=0;i<N*d;i+=d)
            qsort(&all_M[i], d, sizeof(int), cmpfunc);
        clock_t end = clock();
        double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
        printf("elapsed time : %f ms\n",time_spent*1000);
        fprintf(f, "Quicksort,CPU,%f\n",TimeVar);
        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&all_M[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        // ______________________Clean Question 4_____________________________
        // clean copy 
        free(all_M);
        free(all_STM);
        free(all_size_A);
        free(all_size_B);
        testCUDA(hipFree(h_all_M));
        testCUDA(hipFree(h_all_STM));
        testCUDA(hipFree(h_all_size_A));
        testCUDA(hipFree(h_all_size_B));

        // clean zero copy
        testCUDA(hipHostFree(host_all_M));
        testCUDA(hipHostFree(host_all_STM));
        testCUDA(hipHostFree(host_all_size_A));
        testCUDA(hipHostFree(host_all_size_B));
        fclose(f); 
    #endif

    //___________________________ Question 5__________________________________
   
    #if QUESTION == 5
        // We chose to use copy because it's faster than zero copy
        FILE *f = fopen("../results/results5.csv", "w"); 
        fprintf(f, "N,d,time\n");
        // test for several value of N and d
        int Nmax = 1000000;
        if(argc == 2){
            if(atoi(argv[1])<Nmax && atoi(argv[1])> 1000 ) Nmax = atoi(argv[1]);
        }
        for(int N = 10; N<Nmax; N*=10){//10000000 max 
            for (int d = 2; d<=1024; d*=2){
                int* all_M = (int *) malloc(N*d*sizeof(int));
                int* all_STM = (int *) malloc(N*d*sizeof(int));
                int* all_size_A = (int *) malloc(N*sizeof(int));
                int* all_size_B = (int *) malloc(N*sizeof(int));
                int* h_all_M;
                int* h_all_STM;
                int* h_all_size_A;
                int* h_all_size_B;

                // allocation on device for save size
                testCUDA(hipMalloc((void **)&h_all_size_A,N*sizeof(int)));
                testCUDA(hipMalloc((void **)&h_all_size_B,N*sizeof(int)));

                // Initialisation size
                int size_all_A=0;
                int size_all_B=0;
                int sizeA;
                int sizeB;
                for(int i = 0;i<N;i++){ 
                    sizeA = rand()%d+1;
                    sizeB = (d-sizeA);
                    all_size_A[i] = sizeA;
                    all_size_B[i] = sizeB;
                    size_all_A += sizeA;
                    size_all_B +=sizeB;
                }

                // copy of all size on device
                testCUDA(hipMemcpy(h_all_size_A, all_size_A, N*sizeof(int), hipMemcpyHostToDevice));
                testCUDA(hipMemcpy(h_all_size_B, all_size_B, N*sizeof(int), hipMemcpyHostToDevice));
                
                // allocation on device of M and STM
                testCUDA(hipMalloc((void **)&h_all_M,N*d*sizeof(int)));
                testCUDA(hipMalloc((void **)&h_all_STM,N*d*sizeof(int)));

                // Start initialisation of the first arrays A0 and B0
                if(all_size_A[0]!=0){
                    all_M[0]=rand()%20+1;
                    for(int j = 1;j<all_size_A[0];j++){
                        all_M[j]=all_M[j-1]+rand()%20+1;
                    }
                }
                if(all_size_B[0]!=0){
                    all_M[all_size_A[0]]=rand()%20+1;
                    for(int j = all_size_A[0]+1;j<all_size_B[0]+all_size_A[0];j++){
                        all_M[j]=all_M[j-1]+rand()%20+1;
                    }
                }
                int tmp_A=all_size_A[0];
                int tmp_B=all_size_B[0];

                // Initialisation of all arrays 
                for(int i = 1;i<N;i++){ 
                    if(all_size_A[i]!=0){
                        all_M[tmp_A+tmp_B]=rand()%20+1;
                        for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_A[i];j++){
                            all_M[j]=all_M[j-1]+rand()%20+1;
                        }
                        tmp_A+= all_size_A[i];
                
                    }
                    if(all_size_B[i]!=0){
                        all_M[tmp_A+tmp_B]=rand()%20+1;
                        for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_B[i];j++){
                            all_M[j]=all_M[j-1]+rand()%20+1;
                        }
                        tmp_B+= all_size_B[i];
                    }
                }
                // copy all_M on h_all_M on the device
                testCUDA(hipMemcpy(h_all_M, all_M, N*d*sizeof(int), hipMemcpyHostToDevice));

                int numBlocks = N; //big number
                int threadsPerBlock = d; // multiple of d
                testCUDA(hipEventRecord(start));
                mergeSmallBatch_k_shared<<<numBlocks,threadsPerBlock,d*sizeof(int)>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
                testCUDA(hipEventRecord(stop));
                testCUDA(hipEventSynchronize(stop));
                testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
                printf("elapsed time : N = %10d, d = %10d: %f ms\t",N,d,TimeVar);
                fprintf(f, "%d,%d,%f\n",N,d,TimeVar);
                testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

                // _______________Check results_______________
                int all_sorted=1;
                int sorted;
                for(int i = 0;i<N*d;i+=d){
                    sorted = is_sorted(&all_STM[i],d);
                    if(sorted ==0){
                        cout<<"Check sorted : "<<sorted<<endl;
                        all_sorted = 0;
                    }
                }
                if(all_sorted==1){
                    printf("Each array is sorted !\n");
                }
                else{
                    printf("There is an unsorted array !\n");
                }

                free(all_M);
                free(all_STM);
                free(all_size_A);
                free(all_size_B);
                testCUDA(hipFree(h_all_M));
                testCUDA(hipFree(h_all_STM));
                testCUDA(hipFree(h_all_size_A));
                testCUDA(hipFree(h_all_size_B));
            }
        }
        fclose(f); 
    #endif
    
    //___________ Cleaning up ____________________
    #if QUESTION == 2||QUESTION==1
    free(A);
    free(B);
    free(M);
    #endif 
	testCUDA(hipEventDestroy(start));
    testCUDA(hipEventDestroy(stop));
    hipDeviceReset();
	return 0;
}
