#include "hip/hip_runtime.h"
/****************************************************************************
 * Copyright (C) 2020 by Arthur Zucker @ Apavou Clément                     *
 *                                                                          *
 * This file is part of Box.                                                *
 *
 ****************************************************************************/

/**
 * @file main.cu
 * @author Arthur Zucker & Clément Apavou  
 * @date 912 Dec 2020
 * @brief Main file used to produce results for each questions
 *
 * In this porject, we tackled the MERGE SORT problem on GPU
 * using CUDA. We answered questions from a subject. If you want to 
 * see the original Merge sort articles, 
 * @see https://www.researchgate.net/profile/Oded-Green/publication/254462662_GPU_merge_path_a_GPU_merging_algorithm/links/543eeaa00cf2e76f02244884/GPU-merge-path-a-GPU-merging-algorithm.pdf
 * @see https://arxiv.org/pdf/1406.2628.pdf 
 */


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "merge.h"
#include "batch_merge.h"
#include "utils.h"
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))
using namespace std;
texture <int> texture_referenceA ;
texture <int> texture_referenceB ;
#define QUESTION 3  /**< Choose from {1,2,3,4,5} depending on the question */
#define INFO 0      /**< Set to 1 if you need to see GPU infromations. */


int main(int argc, char* argv[]) {
    hipDeviceReset();
    //___________ Basic initialisation ___________
	srand((unsigned int)time(NULL));
	int nDevices;
	hipGetDeviceCount(&nDevices);
    // int Tmax;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        #if INFO == 1
            printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
            printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
            printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
            printf("Max number of threads per multiprocessor : %d\n",prop.maxThreadsPerMultiProcessor);
            printf("Number of multiprocessors on device : %d\n",prop.multiProcessorCount);
            printf("Amount of Shared mem available for int : %d\n",prop.sharedMemPerMultiprocessor/sizeof(int));
            printf("Max running threads : %d\n",prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount);
        #endif
        // Tmax = prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount;
	}
    //Tmax =1024;
	hipSetDevice(0);
    testCUDA(hipSetDeviceFlags(hipDeviceMapHost));
    //____________________________________________

    //___________ Initialising size of arrays  ___________
    #if QUESTION==2 || QUESTION ==1
        int sizeA,sizeB;
        if (argc < 2) {sizeA = rand()%1024;sizeB = rand()%(1024-sizeA);} // If no arguments are provided, set random sizes
        else if(argc == 2){sizeA=atoi(argv[1]);sizeB=atoi(argv[1]);}
        else{sizeA=atoi(argv[1]);sizeB=atoi(argv[2]);}
        int sizeM = sizeA+sizeB;
        printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
        int *hostA,*thostA,*hostB,*thostB,*hostM,*hA,*hB,*hM;
        int *seqM = (int *) malloc(sizeM*sizeof(int));
        int *A = (int *) malloc(sizeA*sizeof(int));
        int *B = (int *) malloc(sizeB*sizeof(int));
        int *M = (int *) malloc(sizeM*sizeof(int));
        A[0]=rand()%20;
        B[0]=rand()%20;
        for(int i=1;i<sizeA;i++){A[i]=A[i-1]+rand()%20+1;}
        for(int i=1;i<sizeB;i++){B[i]=B[i-1]+rand()%20+1;}
    #endif
    #if QUESTION == 5
        int sizeM;
        if (argc < 2) {sizeM = rand()%1024;} 
        if (argc == 2) {sizeM=atoi(argv[1]);} // If no arguments are provided, set random sizes
        printf("|M| = %d\n",sizeM);
    #endif
    //___________________________ Useful time stamps _________________________________
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
    float TimeVar=0;


    //___________________________ Question 1 _________________________________
    #if QUESTION == 1
        //___________ TO DO: explain texture memory ___________
        // Copy 
        testCUDA(hipMalloc((void **)&thostA,sizeA*sizeof(int)));
        testCUDA(hipMalloc((void **)&thostB,sizeB*sizeof(int)));
        testCUDA(hipMalloc((void **)&thostM,sizeM*sizeof(int)));

        testCUDA(hipMemcpy(thostA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
        testCUDA(hipMemcpy(thostB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));
        // texture memory
        testCUDA (hipBindTexture(0,texture_referenceA, thostA,sizeA*sizeof(int)));
        testCUDA (hipBindTexture(0,texture_referenceB, thostB,sizeB*sizeof(int)));
        //____________________________________________
        // Zero copy
        testCUDA(hipHostAlloc(&hostA,sizeA*sizeof(int),hipHostMallocMapped)); //hipHostMallocWriteCombined
        testCUDA(hipHostAlloc(&hostB,sizeB*sizeof(int),hipHostMallocMapped));
        hostA[0]=rand()%20;
        hostB[0]=rand()%20;
        for(int i=1;i<sizeA;i++){hostA[i]=hostA[i-1]+rand()%20+1;}
        for(int i=1;i<sizeB;i++){hostB[i]=hostB[i-1]+rand()%20+1;}

        // WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read
        // efficiently by most CPUs. WC memory is a good option for buffers that will be written by the CPU and read by the device
        // via mapped pinned memory or host->device transfers.

        testCUDA(hipHostAlloc(&hostM,sizeM*sizeof(int),hipHostMallocMapped)); // in order to do zero copy
        /*testCUDA(hipHostGetDevicePointer((void **)&pM, (void *) hostM,0));
        testCUDA(hipHostGetDevicePointer((void **)&pA, (void *) hostA,0));
        testCUDA(hipHostGetDevicePointer((void **)&pB, (void *) hostB,0));
        */
        //_______________ Sequential _________________
        printf("_______________ Sequential _________________\n");
        clock_t begin = clock();
        merged_path_seq(A,B,seqM,sizeA,sizeB);
        clock_t end = clock();
        double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
        printf("elapsed time : %f ms\n",time_spent*1000);
        cout<<"Check sorted : "<<is_sorted(seqM,sizeM)<<endl;
        //____________________________________________


        //_____________________________ Zero copy ______________________________________________________________
        printf("__________________________Zero copy________________________________\n");
        printf("_______________ Zero copy Normal ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

        //____________________________________________

        //___________ Zero copy Shared _________________________
        printf("________________ Zero copy Shared ___________________\n");
        testCUDA(hipEventRecord(start));
        mergeSmall_k_shared<<<1,sizeM,sizeM*sizeof(int)>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        //mergeSmall_k_shared<<<1,sizeM>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

        //____________________________________________

        //___________ Zero copy LDG ____________________________
        printf("_____________________ Zero copy LDG ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k_ldg<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
        //____________________________________________

        //___________ Texture ________________________
        printf("________________ Texture ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
        //____________________________________________

        for(int i=1;i<sizeA;i++){hostM[i]=0;}
        
        //_____________________________ Copy ______________________________________________________________
        printf("__________________________Copy________________________________\n");
        printf("_______________copy Normal ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k<<<1,1024>>>(thostA,thostB,thostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        testCUDA(hipMemcpy(M, thostM, sizeM*sizeof(int), hipMemcpyDeviceToHost)); // retrieve M on the device
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;

        //____________________________________________

        //___________ copy Shared _________________________
        printf("________________copy Shared ___________________\n");
        testCUDA(hipEventRecord(start));
        mergeSmall_k_shared<<<1,sizeM,sizeM*sizeof(int)>>>(thostA,thostB,thostM,sizeA,sizeB,sizeM);
        //mergeSmall_k_shared<<<1,sizeM>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        testCUDA(hipMemcpy(M, thostM, sizeM*sizeof(int), hipMemcpyDeviceToHost));
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;

        //____________________________________________

        //___________ copy LDG ____________________________
        printf("_____________________copy LDG ___________________\n");
        testCUDA(hipEventRecord(start,0));
        mergedSmall_k_ldg<<<1,1024>>>(thostA,thostB,thostM,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        testCUDA(hipMemcpy(M, thostM, sizeM*sizeof(int), hipMemcpyDeviceToHost));
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;
        //____________________________________________

        testCUDA(hipUnbindTexture ( texture_referenceA ));
        testCUDA(hipUnbindTexture ( texture_referenceB ));
        hipFree(thostA);
        hipFree(thostB);
        hipFree(thostM);
        testCUDA(hipHostFree(hostA));
        testCUDA(hipHostFree(hostB));
        testCUDA(hipHostFree(hostM));
    #endif

    //___________________________ Question 2_________________________________
    #if QUESTION==2
        printf("__________________ Path big normal __________________\n");
        int *__restrict__ path;
        int nb_threads = 128;
        int nb_blocks = (sizeM+nb_threads-1)/nb_threads;
        //if(sizeM<1024) nb_blocks=1024;
        int *hA,*hB,*hM;
        testCUDA(hipMalloc((void **)&hA,sizeA*sizeof(int)));
        testCUDA(hipMalloc((void **)&hB,sizeB*sizeof(int)));
        testCUDA(hipMalloc((void **)&hM,sizeM*sizeof(int)));

        testCUDA(hipMemcpy(hA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
        testCUDA(hipMemcpy(hB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));

        testCUDA(hipMalloc((void **)&path,2*(nb_blocks+1)*sizeof(int)));
        testCUDA(hipEventRecord(start,0));
        pathBig_k<<<nb_blocks,nb_threads>>>(hA,hB,path,sizeA,sizeB,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        //____________________________________________
        printf("__________________ Merg big normal _________________\n");
        testCUDA(hipEventRecord(start,0));
        merged_Big_k<<<nb_blocks,nb_threads>>>(hA,hB,hM,path,sizeM);
        testCUDA(hipEventRecord(stop,0));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);
        testCUDA(hipMemcpy(M, hM, sizeM*sizeof(int), hipMemcpyDeviceToHost));
        cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;
        //print_t(hostM,sizeM);
        // printf("__________________ Path big NAIVE __________________\n");
        // testCUDA(hipEventRecord(start,0));
        // pathBig_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,path,sizeA,sizeB,sizeM);
        // testCUDA(hipEventRecord(stop,0));
        // testCUDA(hipEventSynchronize(stop));
        // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        // printf("elapsed time : %f ms\n",TimeVar);
        //____________________________________________
        // printf("__________________ Merg big NAIVE_________________\n");
        // testCUDA(hipEventRecord(start,0));
        // merged_Big_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,hostM,path,sizeM);
        // testCUDA(hipEventRecord(stop,0));
        // testCUDA(hipEventSynchronize(stop));
        // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        // printf("elapsed time : %f ms\n",TimeVar);
        // cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
        //____________________________________________
        //____________________________________________
    #endif

    //___________________________ Question 3_________________________________
    #if QUESTION==3
        int *__restrict__ hD;
        int *__restrict__ hsD;
        int *D  ;
        int *sD ;
        int padding = 0;
        //int nb_threads = 128; // changing it might be smart
        //int nb_blocks = (sizeM+nb_threads-1)/nb_threads;
        printf("__________________ sort M __________________\n");
        int threads_per_blocks = 128;
        FILE *f = fopen("../results/results3.csv", "w"); 
        fprintf(f, "d,time\n");
        for(int d=2;d<262144*2*2;d*=4){
            testCUDA(hipMalloc((void **)&hsD,d*sizeof(int)));
            testCUDA(hipMalloc((void **)&hD ,d*sizeof(int)));
            
            //code to launch on a size != than a power of 2
            if(d != 0 && (d & (d-1)) == 0){
                //printf("|M| is a power of 2\n");
                D  = (int *) malloc(d*sizeof(int));
                sD = (int *) malloc(d*sizeof(int));
                for(int i=0;i<d;i++){D[i]=rand()%d*50+1;}
            }
            else{
                //printf("|M| was not a power of 2, it will be changed\n");
                int power = 1;
                while(power < d) power*=2;
                //printf("new |M| with padding : %d\n",power);
                D  = (int *) malloc(power*sizeof(int));
                sD = (int *) malloc(power*sizeof(int));
                for(int i=0;i<d;i++){D[i]=rand()%d*5+1;}
                for(int i = d;i<power;i++){D[i] = ( int) -1 >> 1;}
                padding = power-d;
                d = power;
            }
            
            // printf("Assigning M\n");  
            testCUDA(hipMemcpy(hD, D, d*sizeof(int), hipMemcpyHostToDevice));
            testCUDA(hipEventRecord(start,0));
            sort_array(hD,hsD,d,threads_per_blocks);
            testCUDA(hipEventRecord(stop,0));
            testCUDA(hipEventSynchronize(stop));
            testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
            printf("d = %10d | t =  %4.10f ms | ",d,TimeVar);
            fprintf(f, "%d,%f\n",d,TimeVar);
            testCUDA(hipMemcpy(sD, hsD, d*sizeof(int), hipMemcpyDeviceToHost));
            cout<<" Sorted : "<<is_sorted(sD,d);
            //____________________Compare with qsort ________________________
            clock_t begin = clock();
            qsort(D, d, sizeof(int), cmpfunc);
            clock_t end = clock();
            double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
            printf("\tquicksort t = %f ms | ",time_spent*1000);
            int sorted = 1;
            for(int i=0;i<d;i++) {
                if(D[i]!=sD[i]){
                    printf("ERROR    i=%d : %d != %d\n",i,D[i],sD[i]);
                    sorted = 0;
                    break;
                }
            }
            if(sorted) printf("arrays are equal\n");
            hipFree(hD);
            hipFree(hsD);
            free(D);
            free(sD);
        }
        fclose(f); 
    #endif
        
    //___________________________ Question 4_________________________________
    #if QUESTION==4
        //_________________________________________ Batch merge part ____________________________________________________
        //___________________________Question 4_________________________________
        // L’objectif est simplement de répartir les block de manière intelligente 
        // sur l’ensemble des calculs Ai + Bi = Mi .
        // N arrays containing Ai and Bi such as |Ai| + |Bi| = d
        // N arrays of size d
        int N = 100; // max 1000000
        int d = 306; 
        
        // ________________________________________Zero Copy______________________________________________ 

        printf("_______________________________Zero copy____________________________________\n");
        int* host_all_M;
        int* host_all_STM;
        int* host_all_size_A;
        int* host_all_size_B;

        // allocation on the device for save all size of Ai and Bi 
        // we choose a 1D representation,  we stocked Ai and Bi in one table M : M = (A1|B1|...|AN|BN) 
        testCUDA(hipHostAlloc(&host_all_size_A,N*sizeof(int),hipHostMallocMapped));
        testCUDA(hipHostAlloc(&host_all_size_B,N*sizeof(int),hipHostMallocMapped));

        // Initialisation of size Ai and Bi such as |Ai| + |Bi| = d 
        int size_all_A=0;
        int size_all_B=0;
        int sizeA;
        int sizeB;
        for(int i = 0;i<N;i++){ 
            sizeA = rand()%d+1;
            sizeB = (d-sizeA);
            host_all_size_A[i] = sizeA;
            host_all_size_B[i] = sizeB;
            size_all_A += sizeA;
            size_all_B +=sizeB;
        }

        // we stocked Ai and Bi in one table M : M = (A1|B1|...|AN|BN) 
        // allocation on device for M and STM of size N*d (N arrays of size d)
        // M will contains N arrays of Ai and Bi not sorted  
        // STM (Sorted M) will contains Mi sorted i.e Ai and Bi merge and sort 
        testCUDA(hipHostAlloc(&host_all_STM,N*d*sizeof(int),hipHostMallocMapped));    
        testCUDA(hipHostAlloc(&host_all_M,N*d*sizeof(int),hipHostMallocMapped));  

        // Start initialisation of the first arrays A0 and B0
        if(host_all_size_A[0]!=0){
            host_all_M[0]=rand()%20+1;
            for(int j = 1;j<host_all_size_A[0];j++){
                host_all_M[j]=host_all_M[j-1]+rand()%20+1;
            }
        }
        if(host_all_size_B[0]!=0){
            host_all_M[host_all_size_A[0]]=rand()%20+1;
            for(int j = host_all_size_A[0]+1;j<host_all_size_B[0]+host_all_size_A[0];j++){
                host_all_M[j]=host_all_M[j-1]+rand()%20+1;
            }
        }
        
        // Initialisation of all arrays 
        int tmp_A=host_all_size_A[0];
        int tmp_B=host_all_size_B[0];
        for(int i = 1;i<N;i++){ 
            if(host_all_size_A[i]!=0){
                host_all_M[tmp_A+tmp_B]=rand()%20+1;
                for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+host_all_size_A[i];j++){
                    host_all_M[j]=host_all_M[j-1]+rand()%20+1;
                }
                tmp_A+= host_all_size_A[i];
        
            }
            if(host_all_size_B[i]!=0){
                host_all_M[tmp_A+tmp_B]=rand()%20+1;
                for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+host_all_size_B[i];j++){
                    host_all_M[j]=host_all_M[j-1]+rand()%20+1;
                }
                tmp_B+= host_all_size_B[i];
            }
        }

        testCUDA(hipEventCreate(&start));
        testCUDA(hipEventCreate(&stop));
        
        printf("_________________ LDG_____________________\n");

        int numBlocks = N; //big number
        int threadsPerBlock = d; // multiple of d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k_ldg<<<numBlocks,threadsPerBlock>>>(host_all_M,host_all_STM,host_all_size_A,host_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);

        // _______________Check results_______________
        int all_sorted=1;
        int sorted;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&host_all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        printf("_________________ Normal_____________________\n");

        numBlocks = N; //big number
        threadsPerBlock = d; // multiple de d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(host_all_M,host_all_STM,host_all_size_A,host_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);

        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&host_all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }
        
        // ________________________________________Copy______________________________________________ 

        printf("__________________________________Copy_______________________________________\n");

        int* all_M = (int *) malloc(N*d*sizeof(int));
        int* all_STM = (int *) malloc(N*d*sizeof(int));
        int* all_size_A = (int *) malloc(N*sizeof(int));
        int* all_size_B = (int *) malloc(N*sizeof(int));
        int* h_all_M;
        int* h_all_STM;
        int* h_all_size_A;
        int* h_all_size_B;

        // allocation on device for save size
        testCUDA(hipMalloc((void **)&h_all_size_A,N*sizeof(int)));
        testCUDA(hipMalloc((void **)&h_all_size_B,N*sizeof(int)));

        // Initialisation size
        size_all_A=0;
        size_all_B=0;
        for(int i = 0;i<N;i++){ 
            sizeA = rand()%d+1;
            sizeB = (d-sizeA);
            all_size_A[i] = sizeA;
            all_size_B[i] = sizeB;
            size_all_A += sizeA;
            size_all_B +=sizeB;
        }

        // copy of all size on device
        testCUDA(hipMemcpy(h_all_size_A, all_size_A, N*sizeof(int), hipMemcpyHostToDevice));
        testCUDA(hipMemcpy(h_all_size_B, all_size_B, N*sizeof(int), hipMemcpyHostToDevice));
        
        // allocation on device of M and STM
        testCUDA(hipMalloc((void **)&h_all_M,N*d*sizeof(int)));
        testCUDA(hipMalloc((void **)&h_all_STM,N*d*sizeof(int)));

        // Start initialisation of the first arrays A0 and B0
        if(all_size_A[0]!=0){
            all_M[0]=rand()%20+1;
            for(int j = 1;j<all_size_A[0];j++){
                all_M[j]=all_M[j-1]+rand()%20+1;
            }
        }
        if(all_size_B[0]!=0){
            all_M[all_size_A[0]]=rand()%20+1;
            for(int j = all_size_A[0]+1;j<all_size_B[0]+all_size_A[0];j++){
                all_M[j]=all_M[j-1]+rand()%20+1;
            }
        }
        tmp_A=all_size_A[0];
        tmp_B=all_size_B[0];

        // Initialisation of all arrays 
        for(int i = 1;i<N;i++){ 
            if(all_size_A[i]!=0){
                all_M[tmp_A+tmp_B]=rand()%20+1;
                for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_A[i];j++){
                    all_M[j]=all_M[j-1]+rand()%20+1;
                }
                tmp_A+= all_size_A[i];
        
            }
            if(all_size_B[i]!=0){
                all_M[tmp_A+tmp_B]=rand()%20+1;
                for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_B[i];j++){
                    all_M[j]=all_M[j-1]+rand()%20+1;
                }
                tmp_B+= all_size_B[i];
            }
        }
        // copy all_M on h_all_M on device
        testCUDA(hipMemcpy(h_all_M, all_M, N*d*sizeof(int), hipMemcpyHostToDevice));

        printf("_________________ LDG_____________________\n");

        numBlocks = N; //big number
        threadsPerBlock = d; // multiple de d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k_ldg<<<numBlocks,threadsPerBlock>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);

        // retrieve STM on device
        testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        printf("_________________ Normal_____________________\n");

        numBlocks = N; //big number
        threadsPerBlock = d; // multiple of d
        testCUDA(hipEventRecord(start));
        mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
        testCUDA(hipEventRecord(stop));
        testCUDA(hipEventSynchronize(stop));
        testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        printf("elapsed time : %f ms\n",TimeVar);

        testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&all_STM[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        // printf("_________________ Shared_____________________\n");

        // numBlocks = N; //big number
        // threadsPerBlock = d; // multiple de d
        // testCUDA(hipEventRecord(start));
        // mergeSmallBatch_k_shared<<<numBlocks,threadsPerBlock>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
        // testCUDA(hipEventRecord(stop));
        // testCUDA(hipEventSynchronize(stop));
        // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
        // printf("elapsed time : %f ms\n",TimeVar);
        // testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

        // // _______________Check results_______________
        // all_sorted=1;
        // for(int i = 0;i<N*d;i+=d){
        //     sorted = is_sorted(&all_STM[i],d);
        //     if(sorted ==0){
        //         cout<<"Check sorted : "<<sorted<<endl;
        //         all_sorted = 0;
        //     }
        // }
        // if(all_sorted==1){
        //     printf("All table are sorted !\n");
        // }
        // else{
        //     printf("There is a table not sorted !\n");
        // }

        // test on quicksort sequential to compare 
        printf("______________________________Quicksort sequential___________________________\n");

        clock_t begin = clock();
        for(int i=0;i<N*d;i+=d)
            qsort(&all_M[i], d, sizeof(int), cmpfunc);
        clock_t end = clock();
        double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
        printf("elapsed time : %f ms\n",time_spent*1000);

        // _______________Check results_______________
        all_sorted=1;
        for(int i = 0;i<N*d;i+=d){
            sorted = is_sorted(&all_M[i],d);
            if(sorted ==0){
                cout<<"Check sorted : "<<sorted<<endl;
                all_sorted = 0;
            }
        }
        if(all_sorted==1){
            printf("All table are sorted !\n");
        }
        else{
            printf("There is a table not sorted !\n");
        }

        // ______________________Clean Question 4_____________________________
        // clean copy 
        free(all_M);
        free(all_STM);
        free(all_size_A);
        free(all_size_B);
        testCUDA(hipFree(h_all_M));
        testCUDA(hipFree(h_all_STM));
        testCUDA(hipFree(h_all_size_A));
        testCUDA(hipFree(h_all_size_B));

        // clean zero copy
        testCUDA(hipHostFree(host_all_M));
        testCUDA(hipHostFree(host_all_STM));
        testCUDA(hipHostFree(host_all_size_A));
        testCUDA(hipHostFree(host_all_size_B));
    #endif

    //___________________________ Question 5__________________________________
   
    #if QUESTION == 5
        // We chose to use copy because it's faster than zero copy
        FILE *f = fopen("../results/results5.csv", "w"); 
        fprintf(f, "N,d,time\n");
        // test for several value of N and d
        for(int N = 10; N<1000000; N*=10){//10000000 max 
            for (int d = 2; d<=1024; d*=2){
                int* all_M = (int *) malloc(N*d*sizeof(int));
                int* all_STM = (int *) malloc(N*d*sizeof(int));
                int* all_size_A = (int *) malloc(N*sizeof(int));
                int* all_size_B = (int *) malloc(N*sizeof(int));
                int* h_all_M;
                int* h_all_STM;
                int* h_all_size_A;
                int* h_all_size_B;

                // allocation on device for save size
                testCUDA(hipMalloc((void **)&h_all_size_A,N*sizeof(int)));
                testCUDA(hipMalloc((void **)&h_all_size_B,N*sizeof(int)));

                // Initialisation size
                int size_all_A=0;
                int size_all_B=0;
                int sizeA;
                int sizeB;
                for(int i = 0;i<N;i++){ 
                    sizeA = rand()%d+1;
                    sizeB = (d-sizeA);
                    all_size_A[i] = sizeA;
                    all_size_B[i] = sizeB;
                    size_all_A += sizeA;
                    size_all_B +=sizeB;
                }

                // copy of all size on device
                testCUDA(hipMemcpy(h_all_size_A, all_size_A, N*sizeof(int), hipMemcpyHostToDevice));
                testCUDA(hipMemcpy(h_all_size_B, all_size_B, N*sizeof(int), hipMemcpyHostToDevice));
                
                // allocation on device of M and STM
                testCUDA(hipMalloc((void **)&h_all_M,N*d*sizeof(int)));
                testCUDA(hipMalloc((void **)&h_all_STM,N*d*sizeof(int)));

                // Start initialisation of the first arrays A0 and B0
                if(all_size_A[0]!=0){
                    all_M[0]=rand()%20+1;
                    for(int j = 1;j<all_size_A[0];j++){
                        all_M[j]=all_M[j-1]+rand()%20+1;
                    }
                }
                if(all_size_B[0]!=0){
                    all_M[all_size_A[0]]=rand()%20+1;
                    for(int j = all_size_A[0]+1;j<all_size_B[0]+all_size_A[0];j++){
                        all_M[j]=all_M[j-1]+rand()%20+1;
                    }
                }
                int tmp_A=all_size_A[0];
                int tmp_B=all_size_B[0];

                // Initialisation of all arrays 
                for(int i = 1;i<N;i++){ 
                    if(all_size_A[i]!=0){
                        all_M[tmp_A+tmp_B]=rand()%20+1;
                        for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_A[i];j++){
                            all_M[j]=all_M[j-1]+rand()%20+1;
                        }
                        tmp_A+= all_size_A[i];
                
                    }
                    if(all_size_B[i]!=0){
                        all_M[tmp_A+tmp_B]=rand()%20+1;
                        for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_B[i];j++){
                            all_M[j]=all_M[j-1]+rand()%20+1;
                        }
                        tmp_B+= all_size_B[i];
                    }
                }
                // copy all_M on h_all_M on the device
                testCUDA(hipMemcpy(h_all_M, all_M, N*d*sizeof(int), hipMemcpyHostToDevice));

                int numBlocks = N; //big number
                int threadsPerBlock = d; // multiple of d
                testCUDA(hipEventRecord(start));
                mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
                testCUDA(hipEventRecord(stop));
                testCUDA(hipEventSynchronize(stop));
                testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
                printf("elapsed time for d = %d: %f ms\n",d,TimeVar);
                fprintf(f, "%d,%d,%f\n",N,d,TimeVar);
                testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

                // _______________Check results_______________
                int all_sorted=1;
                int sorted;
                for(int i = 0;i<N*d;i+=d){
                    sorted = is_sorted(&all_STM[i],d);
                    if(sorted ==0){
                        cout<<"Check sorted : "<<sorted<<endl;
                        all_sorted = 0;
                    }
                }
                if(all_sorted==1){
                    printf("All table are sorted !\n");
                }
                else{
                    printf("There is a table not sorted !\n");
                }

                free(all_M);
                free(all_STM);
                free(all_size_A);
                free(all_size_B);
                testCUDA(hipFree(h_all_M));
                testCUDA(hipFree(h_all_STM));
                testCUDA(hipFree(h_all_size_A));
                testCUDA(hipFree(h_all_size_B));
            }
        }
        fclose(f); 
    #endif

    /**
    * Part 3 : ideads     
   
    */
    //___________ Cleaning up ____________________
    #if QUESTION == 2||QUESTION==1
    free(A);
    free(B);
    free(M);
    #endif 
	testCUDA(hipEventDestroy(start));
    testCUDA(hipEventDestroy(stop));
	return 0;
}
