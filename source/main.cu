#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "merge.h"
#include "batch_merge.h"
#include "utils.h"
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))
using namespace std;
#define TEXTURE 0 //set to 0 to use normal memory, else it will use texture memory for A and B
texture <int> texture_referenceA ;
texture <int> texture_referenceB ;
#define QUESTION 4
#define INFO 0
/*
TO DO :
 - implement using ldg  avec restricted__  et int4 qui contient 4 int, read only memory
     const int* __restrict__  A
 - mergeBig_k
 - pathBig_k

*/
int main(int argc, char* argv[]) {
    hipDeviceReset();
    //___________ Basic initialisation ___________
	srand((unsigned int)time(NULL));
	int nDevices;
	hipGetDeviceCount(&nDevices);
    int Tmax;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        #if INFO == 1
		printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
        printf("Max number of threads per multiprocessor : %d\n",prop.maxThreadsPerMultiProcessor);
        printf("Number of multiprocessors on device : %d\n",prop.multiProcessorCount);
        printf("Amount of Shared mem available for int : %d\n",prop.sharedMemPerMultiprocessor/sizeof(int));
        printf("Max running threads : %d\n",prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount);
        #endif
        Tmax = prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount;
	}
    //Tmax =1024;
	hipSetDevice(0);
    testCUDA(hipSetDeviceFlags(hipDeviceMapHost));
    //____________________________________________

    //___________ Variable declaration ___________
    
    #if QUESTION==2 || QUESTION ==1
    int sizeA,sizeB;
    if (argc < 2) {sizeA = rand()%1024;sizeB = rand()%(1024-sizeA);} // If no arguments are provided, set random sizes
    else if(argc == 2){sizeA=atoi(argv[1]);sizeB=atoi(argv[1]);}
    else{sizeA=atoi(argv[1]);sizeB=atoi(argv[2]);}
    int sizeM = sizeA+sizeB;
    printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
    int *hostA,*thostA,*hostB,*thostB,*hostM,*hA,*hB,*hM;
    int *seqM = (int *) malloc(sizeM*sizeof(int));
    int *A = (int *) malloc(sizeA*sizeof(int));
    int *B = (int *) malloc(sizeB*sizeof(int));
    int *M = (int *) malloc(sizeM*sizeof(int));
    A[0]=rand()%20;
    B[0]=rand()%20;
    for(int i=1;i<sizeA;i++){A[i]=A[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){B[i]=B[i-1]+rand()%20+1;}
    #endif
    #if QUESTION==3
    int sizeM;
    if (argc < 2) {sizeM = rand()%1024;} 
    if (argc == 2) {sizeM=atoi(argv[1]);} // If no arguments are provided, set random sizes
    printf("|M| = %d\n",sizeM);
    #endif
    
    
    


    //___________ call kernels ___________________
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
    float TimeVar=0;



    #if QUESTION == 1
    //___________ TO DO: explain texture memory ___________
    testCUDA(hipMalloc((void **)&thostA,sizeA*sizeof(int)));
    testCUDA(hipMalloc((void **)&thostB,sizeB*sizeof(int)));

    testCUDA(hipMemcpy(thostA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(thostB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));

    testCUDA (hipBindTexture(0,texture_referenceA, thostA,sizeA*sizeof(int)));
    testCUDA (hipBindTexture(0,texture_referenceB, thostB,sizeB*sizeof(int)));
    //____________________________________________
    // zero copy
    testCUDA(hipHostAlloc(&hostA,sizeA*sizeof(int),hipHostMallocMapped)); //hipHostMallocWriteCombined
    testCUDA(hipHostAlloc(&hostB,sizeB*sizeof(int),hipHostMallocMapped));
    hostA[0]=rand()%20;
    hostB[0]=rand()%20;
    for(int i=1;i<sizeA;i++){hostA[i]=hostA[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){hostB[i]=hostB[i-1]+rand()%20+1;}

    // WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read
    // efficiently by most CPUs. WC memory is a good option for buffers that will be written by the CPU and read by the device
    // via mapped pinned memory or host->device transfers.

    testCUDA(hipHostAlloc(&hostM,sizeM*sizeof(int),hipHostMallocMapped)); // in order to do zero copy
    /*testCUDA(hipHostGetDevicePointer((void **)&pM, (void *) hostM,0));
    testCUDA(hipHostGetDevicePointer((void **)&pA, (void *) hostA,0));
    testCUDA(hipHostGetDevicePointer((void **)&pB, (void *) hostB,0));
    */
    //_______________ Sequential _________________
    printf("_______________ Sequential _________________\n");
    clock_t begin = clock();
    merged_path_seq(A,B,seqM,sizeA,sizeB);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("elapsed time : %f ms\n",time_spent*1000);
    cout<<"Check sorted : "<<is_sorted(seqM,sizeM)<<endl;
    //____________________________________________


    
    //____________________________________________

    //___________ Shared _________________________
    printf("________________ Shared ___________________\n");
    testCUDA(hipEventRecord(start));
    mergeSmall_k_shared<<<1,sizeM,sizeM*sizeof(int)>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    //mergeSmall_k_shared<<<1,sizeM>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ texture ________________________
    printf("________________ Texture ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k_texture<<<1,1024>>>(hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ zerocopy _______________________
    printf("_______________ zero copy ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ LDG ____________________________
    printf("_____________________ LDG ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k_ldg<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________
    for(int i=1;i<sizeA;i++){hostM[i]=0;}
    
    #endif
    #if QUESTION==2
    printf("__________________ Path big normal __________________\n");
    int *__restrict__ path;
    int nb_threads = 128;
    int nb_blocks = (sizeM+nb_threads-1)/nb_threads;
    //if(sizeM<1024) nb_blocks=1024;
    testCUDA(hipMalloc((void **)&hA,sizeA*sizeof(int)));
    testCUDA(hipMalloc((void **)&hB,sizeB*sizeof(int)));
    testCUDA(hipMalloc((void **)&hM,sizeM*sizeof(int)));

    testCUDA(hipMemcpy(hA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(hB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));

    testCUDA(hipMalloc((void **)&path,2*(nb_blocks+1)*sizeof(int)));
    testCUDA(hipEventRecord(start,0));
    pathBig_k<<<nb_blocks,nb_threads>>>(hA,hB,path,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
    printf("__________________ Merg big normal _________________\n");
    testCUDA(hipEventRecord(start,0));
    merged_Big_k<<<nb_blocks,nb_threads>>>(hA,hB,hM,path,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    testCUDA(hipMemcpy(M, hM, sizeM*sizeof(int), hipMemcpyDeviceToHost));
    cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;
    //print_t(hostM,sizeM);
    //____________________________________________
    #endif

    #if QUESTION==3
    int *__restrict__ hD;
    int *__restrict__ hsD;
    int *D  ;
    int *sD ;
    int padding = 0;

    if(sizeM != 0 && (sizeM & (sizeM-1)) == 0){
        printf("|M| is a power of 2\n");
        D  = (int *) malloc(sizeM*sizeof(int));
        sD = (int *) malloc(sizeM*sizeof(int));
        for(int i=0;i<sizeM;i++){D[i]=rand()%sizeM*5+1;}
    }
    else{
        printf("|M| was not a power of 2, it will be changed\n");
        int power = 1;
        while(power < sizeM) power*=2;
        printf("new |M| with padding : %d\n",power);
        D  = (int *) malloc(power*sizeof(int));
        sD = (int *) malloc(power*sizeof(int));
        for(int i=0;i<sizeM;i++){D[i]=rand()%sizeM*5+1;}
        for(int i = sizeM;i<power;i++){D[i] = ( int) -1 >> 1;}
        padding = power-sizeM;
        sizeM = power;
    }
    printf("Assigning M\n");
    
    //int nb_threads = 128; // changing it might be smart
    //int nb_blocks = (sizeM+nb_threads-1)/nb_threads;
    printf("__________________ sort M __________________\n");
    
    //if(sizeM<1024) nb_blocks=1024;
    testCUDA(hipMalloc((void **)&hsD,sizeM*sizeof(int)));
    testCUDA(hipMalloc((void **)&hD,sizeM*sizeof(int)));
    testCUDA(hipMemcpy(hD, D, sizeM*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipEventRecord(start,0));
    for(int i=1;i<sizeM;i*=2){
        for(int j=0;j<sizeM;j+=2*i){
            
            if(i>512){
                int *__restrict__ path;
                int nblocks = (2*i+1023)/1024 ;
                //exit(0);
                testCUDA(hipMalloc((void **)&path,2*(nblocks+1)*sizeof(int)));
                pathBig_k   <<<nblocks,1024>>>(&hD[j],&hD[j+i],path,i,i,2*i);
                merged_Big_k<<<nblocks,1024>>>(&hD[j],&hD[j+i],&hsD[j],path,2*i);
            }
            else{
                mergedSmall_k_ldg<<<1,2*i>>>(&hD[j],&hD[j+i],&hsD[j],i,i,2*i);
                //cout<<"Check sorted : "<<is_sorted(&hsD[j],i)<<endl;
                
            }
        }
        int *ht = hD;   
        hD = hsD;
        hsD = ht;
    }
    int *ht = hD;   
    hD = hsD;
    hsD = ht;
    testCUDA(hipMemcpy(sD, hsD, sizeM*sizeof(int), hipMemcpyDeviceToHost));
    //print_t(&sD[padding],sizeM-padding);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    testCUDA(hipMemcpy(sD, hsD, sizeM*sizeof(int), hipMemcpyDeviceToHost));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(&sD[padding],sizeM-padding)<<endl;
    //____________________________________________
    clock_t begin = clock();
    qsort(D, sizeM, sizeof(int), cmpfunc);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("elapsed time : %f ms\n",time_spent*1000);
    cout<<"Check sorted : "<<is_sorted(D,sizeM)<<endl;
    
    #endif
  
    //___________ MergeBig _______________________
    // printf("__________________ Path big sans shared + ldg __________________\n");
    // testCUDA(hipEventRecord(start,0));
    // pathBig_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,path,sizeA,sizeB,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
  
    //___________ Path Big _______________________
    // printf("__________________ Merg big sans shared + ldg _________________\n");
    // testCUDA(hipEventRecord(start,0));
    // merged_Big_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,hostM,path,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    // cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________
    //___________ Cleaning up ____________________
    #if QUESTION == 1
    testCUDA(hipUnbindTexture ( texture_referenceA ));
    testCUDA(hipUnbindTexture ( texture_referenceB ));
    hipFree(thostA);
    hipFree(thostB);
    testCUDA(hipHostFree(hostA));
    testCUDA(hipHostFree(hostB));
    testCUDA(hipHostFree(hostM));
    #endif
    #if QUESTION == 2||QUESTION==1
    free(A);
    free(B);
    free(M);
    #endif 
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
    // ____________________________________________
    #if QUESTION==4
    //__________________________ Batch merge part __________________________
    // L’objectif est simplement de répartir les block de manière intelligente 
    // sur l’ensemble des calculs Ai + Bi = Mi .
    int N = 10; //si trop gros on peut pas allouer sur le gpu (je crois)
    int d = 6; //306

    // _________________________________zero copy____________________________________ 
    printf("_______________________________zero copy____________________________________\n");
    int* host_all_M;
    int* host_all_STM;
    int* host_all_size_A;
    int* host_all_size_B;

    // allocation for save size
    testCUDA(hipHostAlloc(&host_all_size_A,N*sizeof(int),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&host_all_size_B,N*sizeof(int),hipHostMallocMapped));

    // Allocation device 1D
    int size_all_A=0;
    int size_all_B=0;
    int sizeA;
    int sizeB;
    for(int i = 0;i<N;i++){ 
        sizeA = rand()%d+1;
        sizeB = (d-sizeA);
        // printf("|A| = %d, |B| = %d\n",sizeA,sizeB);
        host_all_size_A[i] = sizeA;
        host_all_size_B[i] = sizeB;
        size_all_A += sizeA;
        size_all_B +=sizeB;
    }

    // allocation for M and STM
    printf("size_all_A = %d, size_all_B = %d, size_all_A + size_all_B = %d, size_all_M = %d\n",size_all_A,size_all_B,size_all_A+size_all_B,N*d);
    testCUDA(hipHostAlloc(&host_all_STM,N*d*sizeof(int),hipHostMallocMapped));    
    testCUDA(hipHostAlloc(&host_all_M,N*d*sizeof(int),hipHostMallocMapped));  

    printf("_______ Initialisation___________\n");
    // début init 1D
    if(host_all_size_A[0]!=0){
        host_all_M[0]=rand()%20+1;
        // printf("M[0]=%d\n",host_all_M[0]);
        for(int j = 1;j<host_all_size_A[0];j++){
            host_all_M[j]=host_all_M[j-1]+rand()%20+1;
            // printf("M[%d]=%d\n",j,host_all_M[j]);
        }
    }
    if(host_all_size_B[0]!=0){
        host_all_M[host_all_size_A[0]]=rand()%20+1;
        // printf("M[%d]=%d\n",host_all_size_A[0],host_all_M[host_all_size_A[0]]);
        for(int j = host_all_size_A[0]+1;j<host_all_size_B[0]+host_all_size_A[0];j++){
            host_all_M[j]=host_all_M[j-1]+rand()%20+1;
            // printf("M[%d]=%d\n",j,host_all_M[j]);
        }
    }
    //fin init 1D
    int tmp_A=host_all_size_A[0];
    int tmp_B=host_all_size_B[0];
    for(int i = 1;i<N;i++){ 
        // Initialisation 1D 1 tableau
        if(host_all_size_A[i]!=0){
            host_all_M[tmp_A+tmp_B]=rand()%20+1;
            for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+host_all_size_A[i];j++){
                host_all_M[j]=host_all_M[j-1]+rand()%20+1;
            }
            tmp_A+= host_all_size_A[i];
    
        }
        if(host_all_size_B[i]!=0){
            host_all_M[tmp_A+tmp_B]=rand()%20+1;
            for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+host_all_size_B[i];j++){
                host_all_M[j]=host_all_M[j-1]+rand()%20+1;
            }
            tmp_B+= host_all_size_B[i];
        }
    }

    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop));
    
    printf("_______ Début de la fonction___________\n");
    int numBlocks = N; //big number
    int threadsPerBlock = d; // multiple de d
    testCUDA(hipEventRecord(start));
    mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(host_all_M,host_all_STM,host_all_size_A,host_all_size_B,d);
    testCUDA(hipEventRecord(stop));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);

    printf("_______ Check résultats___________\n");
    int all_sorted=1;
    int sorted;
    for(int i = 0;i<N*d;i+=d){
        sorted = is_sorted(&host_all_STM[i],d);
        if(sorted ==0){
            cout<<"Check sorted : "<<sorted<<endl;
            all_sorted = 0;
        }
    }
    if(all_sorted==1){
        printf("All table are sorted !\n");
    }
    else{
        printf("There is a table not sorted !\n");
    }
    
    // _____________________________________Copy __________________________________________
    printf("_______________________________Copy____________________________________\n");

    int* all_M = (int *) malloc(N*d*sizeof(int));
    int* all_STM = (int *) malloc(N*d*sizeof(int));
    int* all_size_A = (int *) malloc(N*sizeof(int));
    int* all_size_B = (int *) malloc(N*sizeof(int));
    int* h_all_M;
    int* h_all_STM;
    int* h_all_size_A;
    int* h_all_size_B;

    // allocation for save size
    testCUDA(hipMalloc((void **)&h_all_size_A,N*sizeof(int)));
    testCUDA(hipMalloc((void **)&h_all_size_B,N*sizeof(int)));

    // Initialisation size
    size_all_A=0;
    size_all_B=0;
    for(int i = 0;i<N;i++){ 
        sizeA = rand()%d+1;
        sizeB = (d-sizeA);
        // printf("|A| = %d, |B| = %d\n",sizeA,sizeB);
        all_size_A[i] = sizeA;
        all_size_B[i] = sizeB;
        size_all_A += sizeA;
        size_all_B +=sizeB;
    }

    testCUDA(hipMemcpy(h_all_size_A, all_size_A, N*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(h_all_size_B, all_size_B, N*sizeof(int), hipMemcpyHostToDevice));
    

    // allocation for M and STM
    testCUDA(hipMalloc((void **)&h_all_M,N*d*sizeof(int)));
    testCUDA(hipMalloc((void **)&h_all_STM,N*d*sizeof(int)));

    printf("_______ Initialisation___________\n");
    // début init 1D
    if(all_size_A[0]!=0){
        all_M[0]=rand()%20+1;
        // printf("M[0]=%d\n",all_M[0]);
        for(int j = 1;j<all_size_A[0];j++){
            all_M[j]=all_M[j-1]+rand()%20+1;
            // printf("M[%d]=%d\n",j,all_M[j]);
        }
    }
    if(all_size_B[0]!=0){
        all_M[all_size_A[0]]=rand()%20+1;
        // printf("M[%d]=%d\n",all_size_A[0],all_M[all_size_A[0]]);
        for(int j = all_size_A[0]+1;j<all_size_B[0]+all_size_A[0];j++){
            all_M[j]=all_M[j-1]+rand()%20+1;
            // printf("M[%d]=%d\n",j,all_M[j]);
        }
    }
    //fin init 1D
    tmp_A=all_size_A[0];
    tmp_B=all_size_B[0];
    for(int i = 1;i<N;i++){ 
        // Initialisation 1D 1 tableau
        if(all_size_A[i]!=0){
            all_M[tmp_A+tmp_B]=rand()%20+1;
            for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_A[i];j++){
                all_M[j]=all_M[j-1]+rand()%20+1;
            }
            tmp_A+= all_size_A[i];
    
        }
        if(all_size_B[i]!=0){
            all_M[tmp_A+tmp_B]=rand()%20+1;
            for(int j = tmp_A+tmp_B+1;j<tmp_A+tmp_B+all_size_B[i];j++){
                all_M[j]=all_M[j-1]+rand()%20+1;
            }
            tmp_B+= all_size_B[i];
        }
    }
    // for(int i = 1;i<N*d;i++){
    //     printf("M[%d]=%d\n",i,all_M[i]);
    // }
    testCUDA(hipMemcpy(h_all_M, all_M, N*d*sizeof(int), hipMemcpyHostToDevice));

    printf("_______ Début de la fonction___________\n");
    numBlocks = N; //big number
    threadsPerBlock = d; // multiple de d
    testCUDA(hipEventRecord(start));
    mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(h_all_M,h_all_STM,h_all_size_A,h_all_size_B,d);
    testCUDA(hipEventRecord(stop));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    testCUDA(hipMemcpy(all_STM, h_all_STM, N*d*sizeof(int), hipMemcpyDeviceToHost));

    // for(int i = 1;i<N*d;i++){
    //     printf("STM[%d]=%d\n",i,all_STM[i]);
    // }

    printf("_______ Check résultats___________\n");
    all_sorted=1;
    for(int i = 0;i<N*d;i+=d){
        sorted = is_sorted(&all_STM[i],d);
        if(sorted ==0){
            cout<<"Check sorted : "<<sorted<<endl;
            all_sorted = 0;
        }
    }
    if(all_sorted==1){
        printf("All table are sorted !\n");
    }
    else{
        printf("There is a table not sorted !\n");
    }

    printf("_______ Cleaning ___________\n");
    // clean copy 
    free(all_M);
    free(all_STM);
    free(all_size_A);
    free(all_size_B);
    testCUDA(hipFree(h_all_M));
    testCUDA(hipFree(h_all_STM));
    testCUDA(hipFree(h_all_size_A));
    testCUDA(hipFree(h_all_size_B));

    // clean zero copy
    testCUDA(hipHostFree(host_all_M));
    testCUDA(hipHostFree(host_all_STM));
    testCUDA(hipHostFree(host_all_size_A));
    testCUDA(hipHostFree(host_all_size_B));
    
    testCUDA(hipEventDestroy(start));
    testCUDA(hipEventDestroy(stop));
    #endif
	return 0;
}
