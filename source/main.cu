#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "merge.h"
#include "batch_merge.h"
#include "utils.h"
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))
using namespace std;
#define TEXTURE 0 //set to 0 to use normal memory, else it will use texture memory for A and B
texture <int> texture_referenceA ;
texture <int> texture_referenceB ;

/*
TO DO :
 - implement using ldg  avec restricted__  et int4 qui contient 4 int, read only memory
     const int* __restrict__  A
 - mergeBig_k
 - pathBig_k

*/
int main(int argc, char* argv[]) {
    hipDeviceReset();
    //___________ Basic initialisation ___________
	srand((unsigned int)time(NULL));
	int nDevices;
	hipGetDeviceCount(&nDevices);
    int Tmax;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
        printf("Max number of threads per multiprocessor : %d\n",prop.maxThreadsPerMultiProcessor);
        printf("Number of multiprocessors on device : %d\n",prop.multiProcessorCount);
        printf("Amount of Shared mem available for int : %d\n",prop.sharedMemPerMultiprocessor/sizeof(int));
        printf("Max running threads : %d\n",prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount);
        Tmax = prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount;
	}
    Tmax =1024;
	hipSetDevice(0);
    testCUDA(hipSetDeviceFlags(hipDeviceMapHost));
    //____________________________________________

    //___________ Variable declaration ___________
    int sizeA,sizeB;
    if (argc < 3) {sizeA = rand()%1024;sizeB = rand()%(1024-sizeA);} // If no arguments are provided, set random sizes
    else{sizeA=atoi(argv[1]);sizeB=atoi(argv[2]);}
    int sizeM = sizeA+sizeB;
    printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
    int *hostA,*thostA,*hostB,*thostB,*hostM;
    int *seqM = (int *) malloc(sizeM*sizeof(int));
    int *A = (int *) malloc(sizeA*sizeof(int));
    int *B = (int *) malloc(sizeB*sizeof(int));
    A[0]=rand()%20;
    B[0]=rand()%20;
    for(int i=1;i<sizeA;i++){A[i]=A[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){B[i]=B[i-1]+rand()%20+1;}

    //___________ TO DO: explain texture memory ___________
    testCUDA(hipMalloc((void **)&thostA,sizeA*sizeof(int)));
    testCUDA(hipMalloc((void **)&thostB,sizeB*sizeof(int)));

    testCUDA(hipMemcpy(thostA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(thostB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));

    testCUDA (hipBindTexture(0,texture_referenceA, thostA,sizeA*sizeof(int)));
    testCUDA (hipBindTexture(0,texture_referenceB, thostB,sizeB*sizeof(int)));
    //____________________________________________
    // zero copy
    testCUDA(hipHostAlloc(&hostA,sizeA*sizeof(int),hipHostMallocMapped)); //hipHostMallocWriteCombined
    testCUDA(hipHostAlloc(&hostB,sizeB*sizeof(int),hipHostMallocMapped));
    hostA[0]=rand()%20;
    hostB[0]=rand()%20;
    for(int i=1;i<sizeA;i++){hostA[i]=hostA[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){hostB[i]=hostB[i-1]+rand()%20+1;}

    // WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read
    // efficiently by most CPUs. WC memory is a good option for buffers that will be written by the CPU and read by the device
    // via mapped pinned memory or host->device transfers.

    testCUDA(hipHostAlloc(&hostM,sizeM*sizeof(int),hipHostMallocMapped)); // in order to do zero copy
    /*testCUDA(hipHostGetDevicePointer((void **)&pM, (void *) hostM,0));
    testCUDA(hipHostGetDevicePointer((void **)&pA, (void *) hostA,0));
    testCUDA(hipHostGetDevicePointer((void **)&pB, (void *) hostB,0));
    */
    //_______________ Sequential _________________
    printf("_______________ Sequential _________________\n");
    clock_t begin = clock();
    merged_path_seq(A,B,seqM,sizeA,sizeB);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("elapsed time : %f ms\n",time_spent*1000);
    cout<<"Check sorted : "<<is_sorted(seqM,sizeM)<<endl;
    //____________________________________________


    //___________ call kernels ___________________
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
    float TimeVar=0;
    //____________________________________________

    //___________ Shared _________________________
    printf("________________ Shared ___________________\n");
    testCUDA(hipEventRecord(start));
    mergeSmall_k_shared<<<1,sizeM,sizeM*sizeof(int)>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    //mergeSmall_k_shared<<<1,sizeM>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop));
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);

    //____________________________________________

    //___________ texture ________________________
    printf("________________ Texture ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k_texture<<<1,1024>>>(hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ zerocopy _______________________
    printf("_______________ zero copy ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ LDG ____________________________
    printf("_____________________ LDG ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k_ldg<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
	  testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________

    for(int i=1;i<sizeA;i++){hostM[i]=0;}
    //___________ MergeBig _______________________
    printf("__________________ Path big normal __________________\n");
    testCUDA(hipEventRecord(start,0));
    int *__restrict__ path;
    int nb_threads = 163840;
    testCUDA(hipMalloc((void **)&path,sizeM*sizeof(int)));
    pathBig_k<<<nb_threads,1>>>(hostA,hostB,path,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
  
    //___________ Path Big _______________________
    printf("__________________ Merg big normal _________________\n");
    testCUDA(hipEventRecord(start,0));
    merged_Big_k<<<nb_threads,1>>>(hostA,hostB,hostM,path,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //print_t(hostM,sizeM);
    //____________________________________________
  
  
    //___________ MergeBig _______________________
    // printf("__________________ Path big sans shared + ldg __________________\n");
    // testCUDA(hipEventRecord(start,0));
    // pathBig_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,path,sizeA,sizeB,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
  
    //___________ Path Big _______________________
    // printf("__________________ Merg big sans shared + ldg _________________\n");
    // testCUDA(hipEventRecord(start,0));
    // merged_Big_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,hostM,path,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    // cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________
    
    // if |M| >> 163 840 (nb max of threads running at the same time?
    // then, we divide |M|into |M|/163 840 = offset. Each threads takes diag i*offset!
    // Else, if |M|<163 840, then we can use the same amount of threads? Split in diagonals?
    // Thus the dimension of the grid will differ based on the index
    // dim3 block_dim(1,1,1);
    dim3 grid_dim(Tmax,1,1);
    
    
     //___________ MergeBig _______________________
    // printf("__________________ Path big sans shared + ldg __________________\n");
    // testCUDA(hipEventRecord(start,0));
    // pathBig_k<<<grid_dim,block_dim>>>(hostA,hostB,path,sizeA,sizeB,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
  
    //___________ Path Big _______________________
    // printf("__________________ Merg big sans shared + ldg _________________\n");
    // testCUDA(hipEventRecord(start,0));
    // merged_Big_k<<<163840,1>>>(hostA,hostB,hostM,path,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    // cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________
    
    //___________ Cleaning up ____________________
    testCUDA(hipUnbindTexture ( texture_referenceA ));
    testCUDA(hipUnbindTexture ( texture_referenceB ));
    hipFree(thostA);
    hipFree(thostB);
    free(A);
    free(B);
    testCUDA(hipHostFree(hostA));
    testCUDA(hipHostFree(hostB));
	// testCUDA(hipEventDestroy(start));
	// testCUDA(hipEventDestroy(stop));
    testCUDA(hipHostFree(hostM));
    //____________________________________________
    
    //__________________________ Batch merge part __________________________
    int N = 100; //si trop gros on pet pas allouer sur le gpu (je crois)
    int d = 612;
    // int sizeA,sizeB,sizeM;

    // int** all_A = (int**)malloc(N*sizeof(int*));
    // int** all_B = (int**)malloc(N*sizeof(int*));
    // int** all_M = (int**)malloc(N*sizeof(int*));
    // int* all_size_A = (int*)malloc(N*sizeof(int));
    // int* all_size_B = (int*)malloc(N*sizeof(int));
    // int* all_size_M = (int*)malloc(N*sizeof(int));

    int** all_A;
    int** all_B; 
    int** all_M;
    int* all_size_A;
    int* all_size_B;
    // int* all_size_M;
    testCUDA(hipHostAlloc(&all_A,N*sizeof(int*),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_B,N*sizeof(int*),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_M,N*sizeof(int*),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_size_A,N*sizeof(int),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_size_B,N*sizeof(int),hipHostMallocMapped));
    // testCUDA(hipHostAlloc(&all_size_M,N*sizeof(int),hipHostMallocMapped));

    for(int i = 0;i<N;i++){
        sizeA = rand()%d;
        sizeB = (d-sizeA);
        sizeM = sizeA+sizeB;
        // printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
        all_size_A[i] = sizeA;
        all_size_B[i] = sizeB;
        // all_size_M[i] = sizeM;

        // all_A[i] = (int *) malloc(sizeA*sizeof(int));
        // all_B[i] = (int *) malloc(sizeB*sizeof(int));
        // all_M[i] = (int *) malloc(sizeM*sizeof(int));
        testCUDA(hipHostAlloc(&all_A[i],sizeA*sizeof(int),hipHostMallocMapped));
        testCUDA(hipHostAlloc(&all_B[i],sizeB*sizeof(int),hipHostMallocMapped));
        // testCUDA(hipHostAlloc(&all_M[i],sizeM*sizeof(int),hipHostMallocMapped));

        all_A[i][0]=rand()%20;
        all_B[i][0]=rand()%20;
        for(int j=1;j<sizeA;j++){all_A[i][j]=all_A[i][j-1]+rand()%20+1;}
        for(int j=1;j<sizeB;j++){all_B[i][j]=all_B[i][j-1]+rand()%20+1;}
    
    }
    // for(int i = 0;i<N;i++){
    //     printf("size A[%d] = %d\n",i,all_size_A[i]);
    //     printf("size B[%d] = %d\n",i,all_size_B[i]);
    //     //printf("size M[%d] = %d\n",i,all_size_M[i]);
    //     for(int j = 0; j< all_size_A[i];j++){
    //         printf("all_A[%d] = %d\n",j,all_A[i][j]);
    //     }
    // }

    TimeVar=0;
    // testCUDA(hipEventCreate(&start));
	// testCUDA(hipEventCreate(&stop));

    // int numBlocks = 10; 
    // int threadsPerBlock = 1024;
    // testCUDA(hipEventRecord(start,0));
    // mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(all_A,all_B,all_M,all_size_A,all_size_B,d);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);

    //for(int i = 0;i<N;i++){free(all_A[i]);free(all_B[i]);free(all_M[i]);}
    // free(all_A);
    // free(all_B);
    // free(all_M);
    // free(all_size_A);
    // free(all_size_B);
    // free(all_size_M);

    testCUDA(hipHostFree(all_A));
    testCUDA(hipHostFree(all_B));
    testCUDA(hipHostFree(all_M));
    testCUDA(hipHostFree(all_size_A));
    testCUDA(hipHostFree(all_size_B));
    
    testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	return 0;
}
