#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "merge.h"
#include "batch_merge.h"
#include "utils.h"
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))
using namespace std;
#define TEXTURE 0 //set to 0 to use normal memory, else it will use texture memory for A and B
texture <int> texture_referenceA ;
texture <int> texture_referenceB ;
#define QUESTION 4
#define INFO 0
/*
TO DO :
 - implement using ldg  avec restricted__  et int4 qui contient 4 int, read only memory
     const int* __restrict__  A
 - mergeBig_k
 - pathBig_k

*/
int main(int argc, char* argv[]) {
    hipDeviceReset();
    //___________ Basic initialisation ___________
	srand((unsigned int)time(NULL));
	int nDevices;
	hipGetDeviceCount(&nDevices);
    int Tmax;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        #if INFO == 1
		printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
        printf("Max number of threads per multiprocessor : %d\n",prop.maxThreadsPerMultiProcessor);
        printf("Number of multiprocessors on device : %d\n",prop.multiProcessorCount);
        printf("Amount of Shared mem available for int : %d\n",prop.sharedMemPerMultiprocessor/sizeof(int));
        printf("Max running threads : %d\n",prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount);
        #endif
        Tmax = prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount;
	}
    //Tmax =1024;
	hipSetDevice(0);
    testCUDA(hipSetDeviceFlags(hipDeviceMapHost));
    //____________________________________________

    //___________ Variable declaration ___________
    int sizeA,sizeB;
    if (argc < 2) {sizeA = rand()%1024;sizeB = rand()%(1024-sizeA);} // If no arguments are provided, set random sizes
    else if(argc == 2){sizeA=atoi(argv[1]);sizeB=atoi(argv[1]);}
    else{sizeA=atoi(argv[1]);sizeB=atoi(argv[2]);}
    int sizeM = sizeA+sizeB;
    printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
    int *hostA,*thostA,*hostB,*thostB,*hostM,*hA,*hB,*hM;
    int *seqM = (int *) malloc(sizeM*sizeof(int));
    int *A = (int *) malloc(sizeA*sizeof(int));
    int *B = (int *) malloc(sizeB*sizeof(int));
    int *M = (int *) malloc(sizeM*sizeof(int));
    A[0]=rand()%20;
    B[0]=rand()%20;
    for(int i=1;i<sizeA;i++){A[i]=A[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){B[i]=B[i-1]+rand()%20+1;}


    //___________ call kernels ___________________
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
    float TimeVar=0;



    #if QUESTION == 1
    //___________ TO DO: explain texture memory ___________
    testCUDA(hipMalloc((void **)&thostA,sizeA*sizeof(int)));
    testCUDA(hipMalloc((void **)&thostB,sizeB*sizeof(int)));

    testCUDA(hipMemcpy(thostA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(thostB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));

    testCUDA (hipBindTexture(0,texture_referenceA, thostA,sizeA*sizeof(int)));
    testCUDA (hipBindTexture(0,texture_referenceB, thostB,sizeB*sizeof(int)));
    //____________________________________________
    // zero copy
    testCUDA(hipHostAlloc(&hostA,sizeA*sizeof(int),hipHostMallocMapped)); //hipHostMallocWriteCombined
    testCUDA(hipHostAlloc(&hostB,sizeB*sizeof(int),hipHostMallocMapped));
    hostA[0]=rand()%20;
    hostB[0]=rand()%20;
    for(int i=1;i<sizeA;i++){hostA[i]=hostA[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){hostB[i]=hostB[i-1]+rand()%20+1;}

    // WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read
    // efficiently by most CPUs. WC memory is a good option for buffers that will be written by the CPU and read by the device
    // via mapped pinned memory or host->device transfers.

    testCUDA(hipHostAlloc(&hostM,sizeM*sizeof(int),hipHostMallocMapped)); // in order to do zero copy
    /*testCUDA(hipHostGetDevicePointer((void **)&pM, (void *) hostM,0));
    testCUDA(hipHostGetDevicePointer((void **)&pA, (void *) hostA,0));
    testCUDA(hipHostGetDevicePointer((void **)&pB, (void *) hostB,0));
    */
    //_______________ Sequential _________________
    printf("_______________ Sequential _________________\n");
    clock_t begin = clock();
    merged_path_seq(A,B,seqM,sizeA,sizeB);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("elapsed time : %f ms\n",time_spent*1000);
    cout<<"Check sorted : "<<is_sorted(seqM,sizeM)<<endl;
    //____________________________________________


    
    //____________________________________________

    //___________ Shared _________________________
    printf("________________ Shared ___________________\n");
    testCUDA(hipEventRecord(start));
    mergeSmall_k_shared<<<1,sizeM,sizeM*sizeof(int)>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    //mergeSmall_k_shared<<<1,sizeM>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ texture ________________________
    printf("________________ Texture ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k_texture<<<1,1024>>>(hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ zerocopy _______________________
    printf("_______________ zero copy ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;

    //____________________________________________

    //___________ LDG ____________________________
    printf("_____________________ LDG ___________________\n");
    testCUDA(hipEventRecord(start,0));
    mergedSmall_k_ldg<<<1,1024>>>(hostA,hostB,hostM,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________
    for(int i=1;i<sizeA;i++){hostM[i]=0;}
    
    #endif
    #if QUESTION==2
    //___________ MergeBig _______________________
    printf("__________________ Path big normal __________________\n");
    int *__restrict__ path;
    int nb_threads = 5;
    int nb_blocks = (sizeM+nb_threads-1)/nb_threads;
    if(sizeM<1024) nb_blocks=1024;
    nb_blocks = 2;
    testCUDA(hipMalloc((void **)&hA,sizeA*sizeof(int)));
    testCUDA(hipMalloc((void **)&hB,sizeB*sizeof(int)));
    testCUDA(hipMalloc((void **)&hM,sizeM*sizeof(int)));

    testCUDA(hipMemcpy(hA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(hB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));

    testCUDA(hipMalloc((void **)&path,2*(nb_blocks+1)*sizeof(int)));
    testCUDA(hipEventRecord(start,0));
    pathBig_k<<<nb_blocks,nb_threads>>>(hA,hB,path,sizeA,sizeB,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
  
    //___________ Path Big _______________________
    printf("__________________ Merg big normal _________________\n");
    testCUDA(hipEventRecord(start,0));
    merged_Big_k<<<nb_blocks,nb_threads>>>(hA,hB,hM,path,sizeM);
    testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);
    testCUDA(hipMemcpy(M, hM, sizeB*sizeof(int), hipMemcpyDeviceToHost));
    cout<<"Check sorted : "<<is_sorted(M,sizeM)<<endl;
    //print_t(hostM,sizeM);
    //____________________________________________
    #endif
  
    //___________ MergeBig _______________________
    // printf("__________________ Path big sans shared + ldg __________________\n");
    // testCUDA(hipEventRecord(start,0));
    // pathBig_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,path,sizeA,sizeB,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    //____________________________________________
  
    //___________ Path Big _______________________
    // printf("__________________ Merg big sans shared + ldg _________________\n");
    // testCUDA(hipEventRecord(start,0));
    // merged_Big_k_naive_ldg<<<(sizeM+1023)/1024,1024>>>(thostA,thostB,hostM,path,sizeM);
    // testCUDA(hipEventRecord(stop,0));
	// testCUDA(hipEventSynchronize(stop));
    // testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // printf("elapsed time : %f ms\n",TimeVar);
    // cout<<"Check sorted : "<<is_sorted(hostM,sizeM)<<endl;
    //____________________________________________
    //___________ Cleaning up ____________________
    #if QUESTION == 1
    testCUDA(hipUnbindTexture ( texture_referenceA ));
    testCUDA(hipUnbindTexture ( texture_referenceB ));
    hipFree(thostA);
    hipFree(thostB);
    testCUDA(hipHostFree(hostA));
    testCUDA(hipHostFree(hostB));
    testCUDA(hipHostFree(hostM));
    #endif
    free(A);
    free(B);
    free(M);
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
    // ____________________________________________
    #if QUESTION==4
    //__________________________ Batch merge part __________________________
    // L’objectif est simplement de répartir les block de manière intelligente 
    // sur l’ensemble des calculs Ai + Bi = Mi .
    int N = 6; //si trop gros on pet pas allouer sur le gpu (je crois)
    int d = 3; //306
    // int sizeA,sizeB,sizeM;

    // Allocation globale 2D
    // int** all_A = (int**)malloc(N*sizeof(int*));
    // int** all_B = (int**)malloc(N*sizeof(int*));
    // int** all_M = (int**)malloc(N*sizeof(int*));
    // int* all_size_A = (int*)malloc(N*sizeof(int));
    // int* all_size_B = (int*)malloc(N*sizeof(int));
    // int* all_size_M = (int*)malloc(N*sizeof(int));

    // int** all_A;
    // int** all_B; 
    // int** all_M;
    int* all_A;
    int* all_B; 
    int* all_M;
    int* all_size_A;
    int* all_size_B;
    // int* all_size_M;

    // allocation for save size
    testCUDA(hipHostAlloc(&all_size_A,N*sizeof(int),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_size_B,N*sizeof(int),hipHostMallocMapped));
    // testCUDA(hipHostAlloc(&all_size_M,N*sizeof(int),hipHostMallocMapped));
    
    // Allocation device 2D
    // testCUDA(hipHostAlloc(&all_A,N*sizeof(int*),hipHostMallocMapped));
    // testCUDA(hipHostAlloc(&all_B,N*sizeof(int*),hipHostMallocMapped));
    // testCUDA(hipHostAlloc(&all_M,N*sizeof(int*),hipHostMallocMapped));

    // Allocation device 1D
    int size_all_A=0;
    int size_all_B=0;
    for(int i = 0;i<N;i++){ 
        sizeA = rand()%d+1;
        sizeB = (d-sizeA);
        sizeM = sizeA+sizeB;
        printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
        all_size_A[i] = sizeA;
        all_size_B[i] = sizeB;
        size_all_A += sizeA;
        size_all_B +=sizeB;
    }
    // for(int i = 0;i<N;i++){ 
    //     printf("all_size_A[%d]=%d, all_size_B[%d]=%d \n",i,all_size_A[i],i,all_size_B[i]);
    // }
    printf("size_all_A = %d, size_all_B = %d, size_all_A + size_all_B = %d, size_all_M = %d\n",size_all_A,size_all_B,size_all_A+size_all_B,N*d);
    testCUDA(hipHostAlloc(&all_A,size_all_A*sizeof(int),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_B,size_all_B*sizeof(int),hipHostMallocMapped));
    testCUDA(hipHostAlloc(&all_M,N*d*sizeof(int),hipHostMallocMapped));    

    printf("_______ Initialisation___________\n");
    // début init 1D
    all_A[0]=rand()%20;
    for(int j = 1;j<all_size_A[0];j++){
        all_A[j]=all_A[j-1]+rand()%20+1;
    }
    all_B[0]=rand()%20;
    for(int j = 1;j<all_size_B[0];j++){
        all_B[j]=all_B[j-1]+rand()%20+1;
    }
    // fin init 1D
    int tmp_A=all_size_A[0];
    int tmp_B=all_size_B[0];
    for(int i = 1;i<N;i++){ // mettre N pour deux 2D et commencer à 0 
        // printf("i = %d\n",i);
        // sizeA = rand()%d;
        // sizeB = (d-sizeA);
        // sizeM = sizeA+sizeB;
        // printf("|A| = %d, |B| = %d, |M| = %d\n",sizeA,sizeB,sizeM);
        // all_size_A[i] = sizeA;
        // all_size_B[i] = sizeB;
        
        // all_size_M[i] = sizeM;

        // Allocation gobale 2D
        // all_A[i] = (int *) malloc(sizeA*sizeof(int));
        // all_B[i] = (int *) malloc(sizeB*sizeof(int));
        // all_M[i] = (int *) malloc(sizeM*sizeof(int));
        
        // Allocation device 2D
        // testCUDA(hipHostAlloc(&all_A[i],sizeA*sizeof(int),hipHostMallocMapped));
        // testCUDA(hipHostAlloc(&all_B[i],sizeB*sizeof(int),hipHostMallocMapped));
        // testCUDA(hipHostAlloc(&all_M[i],sizeM*sizeof(int),hipHostMallocMapped));

        // Initialisation 2D
        // all_A[i][0]=rand()%20;
        // all_B[i][0]=rand()%20;
        // for(int j=1;j<sizeA;j++){all_A[i][j]=all_A[i][j-1]+rand()%20+1;}
        // for(int j=1;j<sizeB;j++){all_B[i][j]=all_B[i][j-1]+rand()%20+1;}
        
        // Initialisation 1D
        all_A[tmp_A]=rand()%20;
        for(int j = tmp_A+1;j<tmp_A+all_size_A[i];j++){
            all_A[j]=all_A[j-1]+rand()%20+1;
        }
        tmp_A+= all_size_A[i];

        // for(int j = all_size_A[i-1];j<all_size_A[i];j++){
        //     printf("all_A[%d] = %d\n",j,all_A[j]);
        // }

        all_B[tmp_B]=rand()%20;
        for(int j = tmp_B+1;j<tmp_B+all_size_B[i];j++){
            all_B[j]=all_B[j-1]+rand()%20+1;
        }
        tmp_B+= all_size_B[i];
    }
    // for(int i=0;i<N;i++){
    //     printf("all_size_B[%d] = %d\n",i,all_size_B[i]);
    // }
    // for(int i = 0;i<size_all_B;i++){
    //     printf("all_B[%d] = %d\n",i,all_B[i]);
    // }

    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop));
    
    printf("_______ Début de la fonction___________\n");
    int numBlocks = N; //big number
    int threadsPerBlock = d; // multiple de d
    testCUDA(hipEventRecord(start));
    mergeSmallBatch_k<<<numBlocks,threadsPerBlock>>>(all_A,all_B,all_M,all_size_A,all_size_B,N*d,d);
    testCUDA(hipEventRecord(stop));
	testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    printf("elapsed time : %f ms\n",TimeVar);

    printf("_______ Check résultats___________\n");

    for(int i = 0;i<size_all_A;i++){
        printf("all_A[%d] = %d\n",i,all_A[i]);
    }
    for(int i = 0;i<size_all_B;i++){
        printf("all_B[%d] = %d\n",i,all_B[i]);
    }
    for(int i = 0;i<N*d;i++){
        printf("M[%d]=%d\n",i,all_M[i]);
    }

    // for(int i = 0;i<N;i++){
    //     //printf("%d\n",i);
    //     cout<<"Check sorted : "<<is_sorted(all_M[i],d)<<endl;
    // }

    // for(int i = 0;i<N;i++){
    //     //printf("%d\n",i);
    //     cout<<"Check sorted : "<<is_sorted(all_M[i],d)<<endl;
    // }
    
    
    //for(int i = 0;i<N;i++){free(all_A[i]);free(all_B[i]);free(all_M[i]);}
    // free(all_A);
    // free(all_B);
    // free(all_M);
    // free(all_size_A);
    // free(all_size_B);
    // free(all_size_M);
    // for(int i = 0;i<N;i++){testCUDA(hipHostFree(all_A[i]));testCUDA(hipHostFree(all_B[i]));testCUDA(hipHostFree(all_M[i]));}
    // testCUDA(hipHostFree(all_A));
    // testCUDA(hipHostFree(all_B));
    // testCUDA(hipHostFree(all_M));
    // testCUDA(hipHostFree(all_size_A));
    // testCUDA(hipHostFree(all_size_B));
    
    // testCUDA(hipEventDestroy(start));
    // testCUDA(hipEventDestroy(stop));
    #endif
	return 0;
}
