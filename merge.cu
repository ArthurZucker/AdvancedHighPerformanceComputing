#include "hip/hip_runtime.h"
/**************************************************************
This code is an implementation of the merging of two arrays
as describes in the subject
Both the sequential and parralele versions will be detailed in 
order to asses the performances
***************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>  

#define TEXTURE 1 //set to 0 to use normal memory, else it will use texture memory for A and B
// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       printf("%s\n",hipGetErrorString(error));
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))



texture <int> texture_referenceA ;
texture <int> texture_referenceB ;

void merged_path_seq(int **A,int **B, int **M,int a, int b);
__global__ void merged_path_par(int **A,int **B, int **M);
__global__ void mergedSmall_k(int **A,int **B, int **M);
__global__ void mergeSmallBatch_k(int **A,int **B, int **M);
__device__ void pathBig_k(int **A,int **B, int **M);
__device__ void pathBig_k(int **A,int **B, int **M);

int main(int argc, char* argv[]) {
    //___________ Basic initialisation ___________
	srand((unsigned int)time(NULL));
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Max Grid size: %dx%d\n",  prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("Max Thread Dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max Thread per blocks: %d\n", prop.maxThreadsPerBlock);
	}
	hipSetDevice(0);
    //____________________________________________
    
    //___________ Variable declaration ___________
    int sizeA;
    int sizeB;
    
    if (argc < 3) {sizeA = rand()%1024;sizeB = rand()%1024-sizeA;} // If no arguments are provided, set random sizes
    else{sizeA=atoi(argv[1]);sizeB=atoi(argv[2]);}
    printf("|A| = %d, |B| = %d\n",sizeA,sizeB);
    int sizeM = sizeA+sizeB;
    int *hostA;
    int *hostB;
    int *hostM;
    
    
    
    //___________ TO DO: explain texture memory ___________
    #if TEXTURE == 1
    int *A = (int *) malloc(sizeA*sizeof(int));
    int *B = (int *) malloc(sizeB*sizeof(int));
    A[0]=rand()%20;
    B[0]=rand()%20;
    for(int i=1;i<sizeA;i++){A[i]=A[i-1]+rand()%20+1;}
    for(int i=1;i<sizeB;i++){B[i]=B[i-1]+rand()%20+1;}
    
    testCUDA(hipMalloc((void **)&hostA,sizeA*sizeof(int)));
    testCUDA(hipMalloc((void **)&hostB,sizeB*sizeof(int)));
    
    testCUDA(hipMemcpy(hostA, A, sizeA*sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(hostB, B, sizeB*sizeof(int), hipMemcpyHostToDevice));
    

    
    
    testCUDA (hipBindTexture(0,texture_referenceA, hostA,sizeA*sizeof(int)));
    testCUDA (hipBindTexture(0,texture_referenceB, hostB,sizeB*sizeof(int)));
    printf("texture step passed\n");
    #elif
    testCUDA(hipHostAlloc(&hostA,sizeA*sizeof(int),hipHostMallocWriteCombined));
    testCUDA(hipHostAlloc(&hostB,sizeB*sizeof(int),hipHostMallocWriteCombined));
    #endif
        // WC memory can be transferred across the PCI Express bus more quickly on some system configurations, but cannot be read 
        // efficiently by most CPUs. WC memory is a good option for buffers that will be written by the CPU and read by the device 
        // via mapped pinned memory or host->device transfers.
    testCUDA(hipHostAlloc(&hostM,sizeM*sizeof(int),hipHostMallocMapped)); // in order to do zero copy
    // alternative for M : testCUDA(hipMalloc(&hostM,sizeM*sizeof(int)));
    
    
    //____________________________________________
    //___________ Initialize host table ___________
    
    
    // tex1Dfetch()
    
    
    //____________________________________________
    //___________ Cleaning up ____________________
    
    #if TEXTURE == 1
    testCUDA(hipUnbindTexture ( texture_referenceA ));
    testCUDA(hipUnbindTexture ( texture_referenceB ));
    #elif
    testCUDA(hipHostFree(hostA));
    testCUDA(hipHostFree(hostB));
    testCUDA(hipHostFree(hostM));
    #endif
    
    
    //____________________________________________
	return 0;
}

void merged_path_seq(int **A,int **B, int **M,int a, int b){
	int m = a+b;
	int i = 0;
	int j = 0;
	while(i+j<m){
		if(i>=a){
			M[i+j]=B[i];
			j++;}
		else if(j>=m ||A[i]<B[j]){
			M[i+j]=A[i];
			i++;}
		else{
			M[i+j]=B[i];
			j++;}
	}      
	return;
}
__global__ void mergedSmall_k(int **A,int **B, int **M){
    int i = threadIdx.x;
	return;       
}