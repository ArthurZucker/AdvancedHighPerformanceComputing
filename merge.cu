#include "hip/hip_runtime.h"
/**************************************************************
This code is an implementation of the merging of two arrays
as describes in the subject
Both the sequential and parralele versions will be detailed in 
order to asses the performances
***************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>  
#include <iostream>
#include <algorithm>
#include <iterator>
#include <assert.h>
#include <time.h>
#include "merge.h"
using namespace std;
void merged_path_seq(const int *__restrict__ A,const int *__restrict__ B, int *__restrict__ M,const int a, const int b){
	int m = a+b;
	int i = 0;
	int j = 0;
	while(i+j<m){
		if(i>=a){
			M[i+j]=B[j];
			j++;}
		else if(j>=b ||A[i]<B[j]){
			M[i+j]=A[i];
			i++;}
		else{
			M[i+j]=B[j];
			j++;}
	}      
}
__global__ void mergedSmall_k_texture(int *__restrict__ M,const int sA, const int sB, const int sM){
    int i = threadIdx.x;
    //printf("");
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            int AQy_1 = tex1Dfetch( texture_referenceA, (Q.y-1));
            int AQy   = tex1Dfetch( texture_referenceA, Q.y);
            int BQx_1 = tex1Dfetch( texture_referenceB, (Q.x-1));
            int BQx   = tex1Dfetch( texture_referenceB, Q.x);
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || AQy > BQx_1)){
                if(Q.x==sB || Q.y==0 || AQy_1<=BQx){
                   if(Q.y < sA && (Q.x == sB || AQy<=BQx)){
                        M[i] = AQy;
                   }
                   else{
                        M[i] = BQx;
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}
__global__ void mergedSmall_k_ldg(const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ M,int sA, int sB, int sM){
    int i = threadIdx.x;
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            int AQy_1 = __ldg(&A[Q.y-1]);
            int AQy   = __ldg(&A[Q.y]);
            int BQx_1 = __ldg(&B[Q.x-1]);
            int BQx   = __ldg(&B[Q.x]);
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || AQy > BQx_1)){
                if(Q.x==sB || Q.y==0 || AQy_1<=BQx){
                   if(Q.y < sA && (Q.x == sB || AQy<=BQx)){
                        M[i] = AQy;
                   }
                   else{
                        M[i] = BQx;
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}
__global__ void mergedSmall_k_ldg2(const int *__restrict__ A,const int *__restrict__ B, int *__restrict__ M,const int sA, const int sB, const int sM){
    int i = threadIdx.x;
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            // int2 could load Qy1 and Qy but does not work here... illegal acces
            int2 AQ = __ldg((int2 *) &A[Q.y-1]);
            int2 BQ = __ldg((int2 *) &B[Q.x-1]);
            int AQy_1 = AQ.x;
            int AQy   = AQ.y;
            int BQx_1 = BQ.x;
            int BQx   = BQ.y;
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || AQy > BQx_1)){
                if(Q.x==sB || Q.y==0 || AQy_1<=BQx){
                   if(Q.y < sA && (Q.x == sB || AQy<=BQx)){
                        M[i] = AQy;
                   }
                   else{
                        M[i] = BQx;
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}
__global__ void mergedSmall_k(const int *__restrict__ A,const int *__restrict__ B, int *__restrict__ M,const int sA, const int sB, const int sM){
    int i = threadIdx.x;
    if(i<sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            int AQy_1 = A[Q.y-1];
            int AQy   = A[Q.y];
            int BQx_1 = B[Q.x-1];
            int BQx   = B[Q.x];
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || AQy > BQx_1)){
                if(Q.x==sB || Q.y==0 || AQy_1<=BQx){
                   if(Q.y < sA && (Q.x == sB || AQy<=BQx)){
                        M[i] = AQy;
                   }
                   else{
                        M[i] = BQx;
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}

__global__ void pathBig_k (const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ path,const int sA,const int sB,const int sM){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i<=sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            int AQy_1 = A[Q.y-1];
            int AQy   = A[Q.y];
            int BQx_1 = B[Q.x-1];
            int BQx   = B[Q.x];
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || AQy > BQx_1)){
                if(Q.x==sB || Q.y==0 || AQy_1<=BQx){
                   if(Q.y < sA && (Q.x == sB || AQy<=BQx)){
                        path[i] = -Q.y; // 0 means I take A
                   }
                   else{
                        path[i] = Q.x; // 1 means I take B
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}


__global__ void pathBig_k_ldg (const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ path,const int sA,const int sB,const int sM){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i<=sM){
        int2 K;
        int2 P;
        if(i>sA){
            K = {i-sA,sA};
            P = {sA,i-sA};
        }
        else{
            K = {0,i};
            P = {i,0};
        }
        while(1){
            int offset = int(abs(K.y-P.y)/2);
            int2 Q = {K.x+offset,K.y-offset};
            int AQy_1 = __ldg(&A[Q.y-1]);
            int AQy   = __ldg(&A[Q.y]);
            int BQx_1 = __ldg(&B[Q.x-1]);
            int BQx   = __ldg(&B[Q.x]);
            if(Q.y >= 0 && Q.x <= sB && (Q.y == sA || Q.x == 0 || AQy > BQx_1)){
                if(Q.x==sB || Q.y==0 || AQy_1<=BQx){
                   if(Q.y < sA && (Q.x == sB || AQy<=BQx)){
                        path[i] = -Q.y; // 0 means I take A
                   }
                   else{
                        path[i] = Q.x;  // 1 means I take B
                   }
                   break;
                }
                else{
                   K = {Q.x+1,Q.y-1};
                }
            }
            else{
                P = {Q.x-1,Q.y+1};
            }
        }
    }
}

__global__ void    merged_Big_k(const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ M, int *__restrict__ path, const int m){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int p = path[i];
    M[i] = p>0? B[p] : A[-p]; // if path[i] == 1 then M[i] = B[i]
    
}

__global__ void    merged_Big_k_ldg(const int *__restrict__ A,const int *__restrict__ B,int *__restrict__ M, int *__restrict__ path, const int m){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int p = __ldg(&path[i]);
    M[i] = p>0? __ldg(&B[p]) : __ldg(&A[-p]); // if path[i] == 1 then M[i] = B[i]
    
}


